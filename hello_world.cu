#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel() {
	printf("Hello World from GPU!\n");
}

int main() {
	dkernel<<<1,1>>>();
	hipDeviceSynchronize();
	return 0;
}