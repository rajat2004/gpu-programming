#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel() {
	printf("Hello World from GPU! %d\n", threadIdx.x);
}

int main() {
	dkernel<<<1,1024>>>();
	hipDeviceSynchronize();
	return 0;
}