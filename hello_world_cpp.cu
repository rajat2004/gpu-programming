
#include <hip/hip_runtime.h>
#include <iostream>
using namespace std;

__global__ void dkernel() {
    printf("Hello World from GPU!\n");
}

int main() {
    dkernel<<<1,332>>>();
    hipDeviceSynchronize();
    return 0;
}