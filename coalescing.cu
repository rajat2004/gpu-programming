#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 1024
__global__ void dkernel(unsigned *a, unsigned chunksize) {
	unsigned start = chunksize * threadIdx.x;
	for (unsigned nn = start; nn < start + chunksize; ++nn) {
		a[nn]++;
	}
}
int main() {
	unsigned *a, chunksize = 32;
	hipMalloc(&a, sizeof(unsigned) * N);
	dkernel<<<1, N/chunksize>>>(a, chunksize);
	hipDeviceSynchronize();
    return 0;
}
