#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include "kernels.h"

__global__ void per_row_kernel(int *in, int N) {
    int row = threadIdx.x;
    // printf("%d\n", row);
    for(int i=0; i<row; i++) {
        int temp = in[row*N + i];
        in[row*N + i] = in[i*N + row];
        in[i*N + row] = temp;
    }
}
