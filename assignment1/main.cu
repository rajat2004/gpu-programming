#include<stdio.h>
#include<hip/hip_runtime.h>
// #include"kernels.h"
#include<stdlib.h>
#include"CS17B042.cu"

#define N 1000

void print_matrix(int* mat, int n) {
    for(int i=0; i<n; i++) {
        for(int j=0; j<n; j++) {
            printf("%d ", mat[i*n + j]);
        }
        printf("\n");
    }
}

int main() {
    int *hmatrix, *matrix;
    hipMalloc(&matrix, N*N*sizeof(int));
    hmatrix = (int *)malloc(N * N * sizeof(int));

    // Initialize lower triangular matrix
    for(int i=0; i<N; i++) {
        for(int j=0; j<N; j++) {
            if (i>=j)
                hmatrix[i*N + j] = rand()%9 + 1;
            else
                hmatrix[i*N + j] = 0;
        }
    }
    printf("Original matrix:\n");
    print_matrix(hmatrix, N);

    hipMemcpy(matrix, hmatrix, N*N*sizeof(int), hipMemcpyHostToDevice);
    per_row_kernel<<<1,N>>>(matrix, N);
    hipMemcpy(hmatrix, matrix, N*N*sizeof(int), hipMemcpyDeviceToHost);

    printf("\n\nTransformed matrix:\n");

    print_matrix(hmatrix, N);
    return 0;
}