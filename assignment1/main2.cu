#include "hip/hip_runtime.h"
#include <stdio.h>
#include "kernels.h"

void print_matrix(int* mat, int n) {
    for(int i=0; i<n; i++) {
        for(int j=0; j<n; j++) {
            printf("%d ", mat[i*n + j]);
        }
        printf("\n");
    }
}

bool check_if_transpose(int* mat1, int* mat2, int n) {
    for(int i=0; i<n; i++) {
        for(int j=0; j<n; j++) {
            if(mat1[i*n + j] != mat2[j*n + i])
                return false;
        }
    }
    return true;
}

int main()
{
    long long unsigned N,i,j;
    scanf("%llu", &N);
    int* mathost, * matdev, *resmat;
    mathost = (int*)malloc(N * N * sizeof(int));
    resmat = (int*)malloc(N*N*sizeof(int));
    hipMalloc(&matdev, N * N * sizeof(int));


    // print_matrix(mathost, N);

    // Initialize lower triangular matrix
    for(i=0; i<N; i++) {
        for(j=0; j<N; j++) {
            if (i>=j)
                mathost[i*N + j] = rand()%9 + 1;
            else
                mathost[i*N + j] = 0;
        }
    }

    // print_matrix(mathost, N);

    hipMemcpy(matdev, mathost, N * N * sizeof(int),
                hipMemcpyHostToDevice);

    int griddim = ceil((float)N / 1024);
    dim3 block1(32, 32);
    per_row_kernel <<< griddim, block1 >>> (matdev, N);
    hipDeviceSynchronize();

    hipMemcpy(resmat, matdev, N * N * sizeof(int),
	           hipMemcpyDeviceToHost);


	printf("\n");
    // print_matrix(resmat, N);

    printf("%d\n", check_if_transpose(mathost, resmat, N));

	griddim = ceil((float)N * N / 1024 * 32 * 32);
	dim3 grid1(griddim, 32, 32);
	// per_element_kernel << <grid1, 1024 >> > (matdev, N);
	// hipDeviceSynchronize();

	// hipMemcpy(mathost, matdev, N * N * sizeof(int),
	// 	hipMemcpyDeviceToHost);

	// printf("\n");
    // print_matrix(mathost, N);

	griddim = ceil((float)N * N / 1024 * 32);
	dim3 grid2(griddim, 32);
	dim3 block2(32, 32);
	// per_element_kernel_2D << <grid2, block2 >> > (matdev, N);
	// hipDeviceSynchronize();

	// hipMemcpy(mathost, matdev, N * N * sizeof(int),
	// 	hipMemcpyDeviceToHost);

	// printf("\n");
    // print_matrix(mathost, N);

	// for (i = 0; i < N; i++)
	// {
	// 	for (j = i + 1; j < N; j++)
	// 	{
	// 		int temp = mathost[i * N + j];
	// 		mathost[i * N + j] = mathost[j * N + i];
	// 		mathost[j * N + i] = temp;
	// 	}
	// }

	printf("\n");
    // print_matrix(mathost, N);
}