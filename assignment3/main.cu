#include<stdio.h>
#include<hip/hip_runtime.h>
#include<string.h>
#include<stdlib.h>

void print_matrix(int* mat, int rows, int cols) {
    for(int i=0; i<rows; i++) {
        for(int j=0; j<cols; j++) {
            printf("%d ", mat[i*cols + j]);
        }
        printf("\n");
    }
}


__device__ void update(int *data, int n, int row, int col, int x, int op) {
    if(op==-1)
        atomicSub(&data[row*n+col], x);
    else
        atomicAdd(&data[row*n+col], x);
}

__global__ void runQueries(int *data, int m, int n, int **queries, int q) {

}


int main(int argc, char *argv[]) {
    if (argc < 3) {
        printf("Usage: ./a.out <input-file-name> <output-file-name>\n");
        return 0;
    }
    // printf("Opening file\n");
    // printf("%s\n", argv[0]);
    // printf("%s\n", argv[1]);
    // printf("%s\n", argv[2]);
    FILE *in = fopen(argv[1], "r");
    int m,n;
    fscanf(in, "%d %d", &m, &n);
    // printf("%d, %d\n", m,n);

    int *data, *ddata;
    data = (int*)malloc(m*n*sizeof(int));
    hipMalloc(&ddata, m*n*sizeof(int));

    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) {
            fscanf(in, "%d", &data[i*n+j]);
        }
    }

    // print_matrix(data, m, n);

    int q;
    char s[10];              // Assuming every number is less than 1000000000

    fscanf(in, "%d", &q);

    int* queries[q];

    int* dqueries[q];
    // int** dqueries;
    // cudaMalloc(&dqueries, q*sizeof(int*));

    for(int i=0; i<q; i++) {
        fscanf(in, "%s", s);
        if (strcmp(s, "U")!=0) {
            // First char is not "U", some problem
            printf("Incorrect input, first character in query must be U, exiting!\n");
            return -1;
        }

        fscanf(in, "%s", s);
        int col = atoi(&s[1]);      // Skip first char C

        fscanf(in, "%s", s);
        int x = atoi(s);            // Value to be matched against column

        fscanf(in, "%s", s);
        int p = atoi(s);            // No of update ops

        int len = p*3 + 3;          // +1 for column, +1 for key, +1 for no. of updates

        queries[i] = (int*)malloc(len*sizeof(int));

        // cudaMalloc(&dqueries[i], len*sizeof(int));
        int *dquery;
        hipMalloc(&dquery, len*sizeof(int));

        queries[i][0] = col;
        queries[i][1] = x;
        queries[i][2] = p;

        for(int j=3; j<len; j+=3) {
            fscanf(in, "%s", s);
            queries[i][j] = atoi(&s[1]);

            fscanf(in, "%s", s);
            queries[i][j+1] = atoi(s);

            fscanf(in, "%s", s);
            queries[i][j+2] = (strcmp(s,"+") ? -1 : 1);     // -1 if -, 1 for +

            // printf("%d %d %d\n", queries[i][j], queries[i][j+1], queries[i][j+2]);
        }

        // cudaMemcpy(dqueries[i], queries[i], len*sizeof(int), cudaMemcpyHostToDevice);
        hipMemcpy(dquery, queries[i], len*sizeof(int), hipMemcpyHostToDevice);
        dqueries[i] = dquery;
    }

    for(int i=0; i<q; i++) {
        int len = queries[i][2]*3 + 3;
        // printf("%d\n", len);
        for(int j=0; j<len; j++) {
            printf("%d ", queries[i][j]);
        }
        printf("\n");
    }

    // Copy Database to GPU
    hipMemcpy(ddata, data, m*n*sizeof(int), hipMemcpyHostToDevice);

    // Copy array of pointers(pointing to queries) to GPU
    int** dquerieslist;
    hipMalloc(&dquerieslist, q*sizeof(int*));

    hipMemcpy(dquerieslist, dqueries, q*sizeof(int*), hipMemcpyHostToDevice);

    // One query per thread
    int n_blocks = ceil((float)q / 1024);

    runQueries<<<n_blocks, 1024>>>(ddata, m, n, dquerieslist, q);

    fclose(in);
    return 0;
}
