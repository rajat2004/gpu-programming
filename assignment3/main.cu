#include<stdio.h>
#include<hip/hip_runtime.h>
#include<string.h>
#include<stdlib.h>

void print_matrix(int* mat, int rows, int cols) {
    for(int i=0; i<rows; i++) {
        for(int j=0; j<cols; j++) {
            printf("%d ", mat[i*cols + j]);
        }
        printf("\n");
    }
}

void print_matrix_file(FILE* f, int* mat, int rows, int cols) {
    for(int i=0; i<rows; i++) {
        for(int j=0; j<cols; j++) {
            fprintf(f, "%d ", mat[i*cols + j]);
        }
        fprintf(f, "\n");
    }
}


// Run query on the specified row
__device__ void runQuery(int *data, int n, int *query, int row) {
    int n_queries = query[2];
    int len = n_queries*3 + 3;

    for(int i=3; i<len; i+=3) {
        int op = query[i+2];

        if (op == -1)
            atomicSub(&data[row*n+query[i]-1], query[i+1]);
        else
            atomicAdd(&data[row*n+query[i]-1], query[i+1]);
    }
}

__global__ void searchQuery(int* data, int n, int m, int* query, int col, int x) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if (tid < m) {
        if(data[tid*n + col-1] == x)
            runQuery(data, n, query, tid);
    }
}

__global__ void runQueries(int* data, int m, int n, int** queries, int q) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;

    if (tid < q) {
        int *query = queries[tid];

        int col = query[0];
        int x = query[1];

        // Search in database
        for(int row=0; row<m; row++) {
            if(data[row*n + col-1] == x)
                runQuery(data, n, query, row);
        }

        // Try kernel in kernel
        // int n_blocks = ceil((float)m / 1024);
        // searchQuery<<<n_blocks, 1024>>>(data, n, m, query, col, x);
    }
}


int main(int argc, char *argv[]) {
    if (argc < 3) {
        printf("Usage: ./a.out <input-file-name> <output-file-name>\n");
        return 0;
    }

    FILE *in = fopen(argv[1], "r");
    if (in == NULL) {
        printf("Error opening input file!\n");
        return -1;
    }

    int m,n;
    fscanf(in, "%d %d", &m, &n);

    int *data, *ddata;
    data = (int*)malloc(m*n*sizeof(int));
    hipMalloc(&ddata, m*n*sizeof(int));

    for(int i=0; i<m; i++) {
        for(int j=0; j<n; j++) {
            fscanf(in, "%d", &data[i*n+j]);
        }
    }

    // print_matrix(data, m, n);

    int q;
    char s[10];                     // Assuming every number is less than 1000000000

    fscanf(in, "%d", &q);

    int* queries[q];                // Storage on CPU

    int* dqueries[q];               // Storage on GPU

    for(int i=0; i<q; i++) {
        fscanf(in, "%s", s);
        if (strcmp(s, "U")!=0) {
            // First char is not "U", some problem
            printf("Incorrect input, first character in query must be U, exiting!\n");
            return -1;
        }

        fscanf(in, "%s", s);
        int col = atoi(&s[1]);      // Skip first char C

        fscanf(in, "%s", s);
        int x = atoi(s);            // Value to be matched against column

        fscanf(in, "%s", s);
        int p = atoi(s);            // No of update ops

        int len = p*3 + 3;          // +1 for column, +1 for key, +1 for no. of updates

        queries[i] = (int*)malloc(len*sizeof(int));

        // For copying query to GPU
        int *dquery;
        hipMalloc(&dquery, len*sizeof(int));

        queries[i][0] = col;
        queries[i][1] = x;
        queries[i][2] = p;

        for(int j=3; j<len; j+=3) {
            fscanf(in, "%s", s);
            queries[i][j] = atoi(&s[1]);

            fscanf(in, "%s", s);
            queries[i][j+1] = atoi(s);

            fscanf(in, "%s", s);
            queries[i][j+2] = (strcmp(s,"+") ? -1 : 1);     // -1 if -, 1 for +

            // printf("%d %d %d\n", queries[i][j], queries[i][j+1], queries[i][j+2]);
        }

        hipMemcpy(dquery, queries[i], len*sizeof(int), hipMemcpyHostToDevice);
        dqueries[i] = dquery;
    }

    // for(int i=0; i<q; i++) {
    //     int len = queries[i][2]*3 + 3;
    //     // printf("%d\n", len);
    //     for(int j=0; j<len; j++) {
    //         printf("%d ", queries[i][j]);
    //     }
    //     printf("\n");
    // }

    // Copy Database to GPU
    hipMemcpy(ddata, data, m*n*sizeof(int), hipMemcpyHostToDevice);

    // Copy array of pointers(pointing to queries) to GPU
    int** dquerieslist;
    hipMalloc(&dquerieslist, q*sizeof(int*));

    hipMemcpy(dquerieslist, dqueries, q*sizeof(int*), hipMemcpyHostToDevice);

    // One query per thread
    int n_blocks = ceil((float)q / 1024);
    runQueries<<<n_blocks, 1024>>>(ddata, m, n, dquerieslist, q);

    hipMemcpy(data, ddata, m*n*sizeof(int), hipMemcpyDeviceToHost);

    // print_matrix(data, m, n);

    // Output to file
    FILE *out = fopen(argv[2], "w");
    if (out == NULL) {
        printf("Error opening output file!");
        return -1;
    }
    print_matrix_file(out, data, m, n);

    fclose(in);
    fclose(out);

    // Free allocated memory
    free(data);
    hipFree(ddata);

    for(int i=0; i<q; i++) {
        free(queries[i]);
        hipFree(dqueries[i]);
    }

    return 0;
}
