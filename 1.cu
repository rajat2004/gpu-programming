
#include <hip/hip_runtime.h>
#include <iostream>
#define N 1024

using namespace std;

__global__ void fun(int* arr) {
    int id = threadIdx.x;
    arr[id] = id*id*id;
}

int main() {
    int ha[N], *a;
    hipMalloc(&a, N*sizeof(N));
    fun<<<1,N>>>(a);
    hipMemcpy(ha, a, N*sizeof(int), hipMemcpyDeviceToHost);
    // cudaDeviceSynchronize();
    for(int i=0; i<N; i++) {
        cout << ha[i] << endl;
    }
    return 0;
}