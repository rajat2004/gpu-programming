#include<stdio.h>
#include<hip/hip_runtime.h>

#define BLOCKSIZE 1024

__global__ void initialize(unsigned* matrix, unsigned N) {
    unsigned id = threadIdx.x * blockDim.y + threadIdx.y;
    matrix[id] = id;
}

__global__ void square_v1(unsigned* matrix, unsigned* result, unsigned N) {
    
}

int main(int nn, char *str[]) {
    unsigned N = atoi(str[1]);
    unsigned *hmatrix, *matrix;

    dim3 block(N, N, 1);

    hipMalloc(&matrix, N*N*sizeof(unsigned));
    hmatrix = (unsigned*)malloc(N*N*sizeof(unsigned));

    // unsigned nblocks = ceil((float)N/BLOCKSIZE);
    // printf("nblocks = %d\n", nblocks);

    initialize<<<1, block>>>(matrix, N);
    hipMemcpy(hmatrix, matrix, N*N*sizeof(unsigned), hipMemcpyDeviceToHost);

    for(int i=0; i<N; i++) {
        for(int j=0; j<N; j++) {
            printf("%4d ", hmatrix[i*N + j]);
        }
        printf("\n");
    }
    return 0;
}
