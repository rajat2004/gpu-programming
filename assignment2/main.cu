#include<stdio.h>
#include<limits.h>
#include<stdlib.h>
#include"hip/hip_runtime.h"


void print_matrix(int* mat, int rows, int cols) {
    for(int i=0; i<rows; i++) {
        for(int j=0; j<cols; j++) {
            printf("%d ", mat[i*cols + j]);
        }
        printf("\n");
    }
}


__global__ void sumRandC(int* mat, int n, int m, int k) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int index = tid*k;

    // printf("%d\n", index);

    int orig_row = index / m;
    int orig_col = index % m;

    int last_row = n*(m+1);
    int row=orig_row, col=orig_col;

    if (orig_row < n) {
        for(int i=0; i<k; i++) {
            int oc_plus_i = orig_col+i;
            
            col = oc_plus_i%m;
            row = orig_row + (oc_plus_i/m);

            // printf("%d %d\n", row, col);

            int val = mat[row*(m+1)+col];
            atomicAdd(&mat[row*(m+1) + m], val);
            atomicAdd(&mat[last_row + col], val);
        }
    }
}

// Min value to be added to each element
__device__ int min_el = INT_MAX;

__global__ void findMin(int* mat, int n, int m, int k) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int orig_index = tid*k;
    int val = INT_MAX;
    int index = orig_index;
    int thread_min = INT_MAX;

    for (int i=0; i<k; i++) {
        index = orig_index + i;

        if (index < n) {
            // Check in last col of each row
            val = mat[index*(m+1) + m];
        }
        else if (index < n+m) {
            // Check in last row
            val = mat[n*(m+1) + (index-n)];
        }
        else
            return;

        if (thread_min > val)
            thread_min = val;
    }
    atomicMin(&min_el, thread_min);
}

__global__ void updateMin(int* mat, int n, int m, int k) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int index = tid*k;

    int orig_row = index / m;
    int orig_col = index % m;

    int row=orig_row, col=orig_col;

    if (row < n) {
        for(int i=0; i<k; i++) {
            int oc_plus_i = orig_col+i;
            
            col = oc_plus_i%m;
            row = orig_row + (oc_plus_i/m);

            mat[row*(m+1)+col]+=min_el;
        }
    }

    if(tid==0) {
        mat[n*(m+1)+m] = min_el;
    }
}


int main() {
    int n,m,k;
    scanf("%d %d %d", &n, &m, &k);
    int *mat, *dmat;
    mat = (int*)calloc((n+1)*(m+1), sizeof(int));

    hipMalloc(&dmat, (n+1)*(m+1)*sizeof(int));


    for(int i=0; i<n; i++) {
        int row = i*(m+1);
        for(int j=0; j<m; j++) {
            scanf("%d", &mat[row+j]);
        }
    }

    // Initialize matrix
    // for(int i=0; i<n; i++) {
    //     for(int j=0; j<m; j++) {
    //         mat[i*(m+1) + j] = rand()%9 + 1;
    //     }
    // }


    hipMemcpy(dmat, mat, (n+1)*(m+1)*sizeof(int), hipMemcpyHostToDevice);
    int gridDim = ceil((float)(n*m) / (1024*k) );
    sumRandC<<<gridDim, 1024>>>(dmat, n, m, k);

    hipDeviceSynchronize();

    gridDim = ceil((float)(n+m)/(1024*k));
    findMin<<<gridDim, 1024>>>(dmat, n, m, k);

    hipDeviceSynchronize();

    gridDim = ceil((float)(n*m) / (1024*k) );
    updateMin<<<gridDim, 1024>>>(dmat, n, m, k);

    hipMemcpy(mat, dmat, (n+1)*(m+1)*sizeof(int), hipMemcpyDeviceToHost);

    print_matrix(mat,n+1,m+1);
    return 0;
}
