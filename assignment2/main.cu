#include<stdio.h>
#include<limits.h>
#include<stdlib.h>
#include"hip/hip_runtime.h"

#ifndef min
    #define min(a,b) ((a)<(b) ? (a):(b))
#endif

void print_matrix(int* mat, int rows, int cols) {
    for(int i=0; i<rows; i++) {
        for(int j=0; j<cols; j++) {
            printf("%d ", mat[i*cols + j]);
        }
        printf("\n");
    }
}

void cpu_func(int* mat, int* res, int n, int m, int k=1) {
    int min_el = INT_MAX;
    int last_row = n*(m+1);
    for(int i=0; i<n; i++) {
        int row = i*(m+1);
        for(int j=0; j<m; j++) {
            res[row + m] += mat[row+j];
            res[last_row + j] += mat[row+j];
        }
        min_el = min(min_el, res[row+m]);
    }
}

int main() {
    int n,m,k;
    scanf("%d %d %d", &n, &m, &k);
    int *mat, *res;
    mat = (int*)calloc((n+1)*(m+1), sizeof(int));
    res = (int*)calloc((n+1)*(m+1), sizeof(int));

    for(int i=0; i<n; i++) {
        int row = i*(m+1);
        for(int j=0; j<m; j++) {
            scanf("%d", &mat[row+j]);
            // res[]
        }
    }

    memcpy(res, mat, (n+1)*(m+1)*sizeof(int));
    cpu_func(mat, res, n, m);

    printf("\n");
    print_matrix(res,n+1,m+1);
    return 0;
}
