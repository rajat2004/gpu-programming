#include<stdio.h>
#include<limits.h>
#include<stdlib.h>
#include"hip/hip_runtime.h"

#ifndef min
    #define min(a,b) ((a)<(b) ? (a):(b))
#endif

void print_matrix(int* mat, int rows, int cols) {
    for(int i=0; i<rows; i++) {
        for(int j=0; j<cols; j++) {
            printf("%d ", mat[i*cols + j]);
        }
        printf("\n");
    }
}

int cpu_func(int* mat, int* res, int n, int m, int k=1) {
    int min_el = INT_MAX;
    int last_row = n*(m+1);
    for(int i=0; i<n; i++) {
        int row = i*(m+1);
        for(int j=0; j<m; j++) {
            res[row + m] += mat[row+j];
            res[last_row + j] += mat[row+j];
        }
        min_el = min(min_el, res[row+m]);
    }
    for(int j=0; j<m; j++) {
        min_el = min(min_el, res[last_row+j]);
    }
    // res[last_row+m] = min_el;
    return min_el;
}


__global__ void sumRandC(int* mat, int n, int m, int k) {
    int threadId1 = blockIdx.x * blockDim.x + threadIdx.x;
    int threadId = threadId1*k;

    // printf("%d\n", threadId);

    int orig_row = threadId / m;
    int orig_col = threadId % m;

    int last_row = n*(m+1);
    int row=orig_row, col=orig_col;

    if (orig_row < n) {
        for(int i=0; i<k; i++) {
            int oc_plus_i = orig_col+i;
            
            col = oc_plus_i%m;
            row = orig_row + (oc_plus_i/m);

            // printf("%d %d\n", row, col);

            int val = mat[row*(m+1)+col];
            atomicAdd(&mat[row*(m+1) + m], val);
            atomicAdd(&mat[last_row + col], val);
        }
    }
}

// Min value to be added to each element
__device__ int min_el = INT_MAX;

__global__ void findMin(int* mat, int n, int m) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int val = INT_MAX;

    if(tid < n) {
        // Check in last col of each row
        val = mat[tid*(m+1)+m];
    }
    else if (tid < n+m) {
        // Check in last row
        val = mat[n*(m+1) + (tid-n)];
    }
    else
        return;

    if (min_el > val)
        atomicMin(&min_el, val);

}

__global__ void updateMin(int* mat, int rows, int cols, int k) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int index = tid*k;

    if (index < rows*cols) {
        for(int i=0; i<k; i++) {
            mat[index+i]+=min_el;
        }
    }
}


int main() {
    int n,m,k;
    scanf("%d %d %d", &n, &m, &k);
    int *mat, *res, *dmat;
    mat = (int*)calloc((n+1)*(m+1), sizeof(int));
    res = (int*)calloc((n+1)*(m+1), sizeof(int));
    hipMalloc(&dmat, (n+1)*(m+1)*sizeof(int));
    // cudaMalloc(&min_el, sizeof(int));

    for(int i=0; i<n; i++) {
        int row = i*(m+1);
        for(int j=0; j<m; j++) {
            scanf("%d", &mat[row+j]);
        }
    }

    // Initialize matrix
    // for(int i=0; i<n; i++) {
    //     for(int j=0; j<m; j++) {
    //         mat[i*(m+1) + j] = rand()%9 + 1;
    //     }
    // }

    // memcpy(res, mat, (n+1)*(m+1)*sizeof(int));
    // cpu_func(mat, res, n, m);
    hipMemcpy(dmat, mat, (n+1)*(m+1)*sizeof(int), hipMemcpyHostToDevice);
    int gridDim = ceil((float)(n*m) / (1024*k) );
    // printf("%d\n", gridDim);
    sumRandC<<<gridDim, 1024>>>(dmat, n, m, k);

    hipDeviceSynchronize();

    gridDim = ceil((float)(n+m)/1024);
    findMin<<<gridDim, 1024>>>(dmat, n, m);

    hipDeviceSynchronize();

    gridDim = ceil((float)((n+1)*(m+1)) / (1024*k) );
    updateMin<<<gridDim, 1024>>>(dmat, n+1, m+1, k);

    hipMemcpy(res, dmat, (n+1)*(m+1)*sizeof(int), hipMemcpyDeviceToHost);

    // printf("\n");
    print_matrix(res,n+1,m+1);
    return 0;
}
