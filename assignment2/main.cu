#include<stdio.h>
#include<limits.h>
#include<stdlib.h>
#include"hip/hip_runtime.h"

#ifndef min
    #define min(a,b) ((a)<(b) ? (a):(b))
#endif

void print_matrix(int* mat, int rows, int cols) {
    for(int i=0; i<rows; i++) {
        for(int j=0; j<cols; j++) {
            printf("%d ", mat[i*cols + j]);
        }
        printf("\n");
    }
}

// void find_min_and_print_matrix(int* mat, int rows, int cols) {
//     int min_el = INT_MAX;
//     for(int i=0; i<rows; i++) {
//         for(int j=0; j<cols; j++) {
//             printf("%d ", mat[i*cols + j]);
//         }
//         printf("\n");
//     }
// }

void cpu_func(int* mat, int* res, int n, int m, int k=1) {
    int min_el = INT_MAX;
    int last_row = n*(m+1);
    for(int i=0; i<n; i++) {
        int row = i*(m+1);
        for(int j=0; j<m; j++) {
            res[row + m] += mat[row+j];
            res[last_row + j] += mat[row+j];
        }
        min_el = min(min_el, res[row+m]);
    }
    for(int j=0; j<m; j++) {
        min_el = min(min_el, res[last_row+j]);
    }
    res[last_row+m] = min_el;
}


__global__ void sumRandC(int* mat, int n, int m, int k) {
    int threadId1 = blockIdx.x * blockDim.x + threadIdx.x;
    int threadId = threadId1*k;

    // printf("%d\n", threadId);

    int orig_row = threadId / m;
    int orig_col = threadId % m;

    int last_row = n*(m+1);
    int row=orig_row, col=orig_col;

    if (orig_row < n) {
        for(int i=0; i<k; i++) {
            int oc_plus_i = orig_col+i;
            
            col = oc_plus_i%m;
            row = orig_row + (oc_plus_i/m);
            // if (oc_plus_i>=m) {
            //     row = orig_row + (oc_plus_i/m);
            //     // col = oc_plus_i%m;
            // } 
            // else {
            //     col = oc_plus_i;
            // }

            printf("%d %d\n", row, col);

            int val = mat[row*(m+1)+col];
            atomicAdd(&mat[row*(m+1) + m], val);
            atomicAdd(&mat[last_row + col], val);
        }
    }
}

int main() {
    int n,m,k;
    scanf("%d %d %d", &n, &m, &k);
    int *mat, *res, *dmat;
    mat = (int*)calloc((n+1)*(m+1), sizeof(int));
    res = (int*)calloc((n+1)*(m+1), sizeof(int));
    hipMalloc(&dmat, (n+1)*(m+1)*sizeof(int));

    for(int i=0; i<n; i++) {
        int row = i*(m+1);
        for(int j=0; j<m; j++) {
            scanf("%d", &mat[row+j]);
        }
    }

    // Initialize matrix
    // for(int i=0; i<n; i++) {
    //     for(int j=0; j<m; j++) {
    //         mat[i*(m+1) + j] = rand()%9 + 1;
    //     }
    // }

    // memcpy(res, mat, (n+1)*(m+1)*sizeof(int));
    // cpu_func(mat, res, n, m);
    hipMemcpy(dmat, mat, (n+1)*(m+1)*sizeof(int), hipMemcpyHostToDevice);
    int gridDim = ceil((float)(n*m) / (1024*k) );
    printf("%d\n", gridDim);
    sumRandC<<<gridDim,1024>>>(dmat, n, m, k);

    hipMemcpy(res, dmat, (n+1)*(m+1)*sizeof(int), hipMemcpyDeviceToHost);

    printf("\n");
    print_matrix(res,n+1,m+1);
    return 0;
}
