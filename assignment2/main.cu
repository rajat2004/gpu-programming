#include<stdio.h>
#include<limits.h>
#include<stdlib.h>
#include"hip/hip_runtime.h"


void print_matrix(int* mat, int rows, int cols) {
    for(int i=0; i<rows; i++) {
        for(int j=0; j<cols; j++) {
            printf("%d ", mat[i*cols + j]);
        }
        printf("\n");
    }
}


__global__ void sumRandC(int* mat, int n, int m, int k) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    int index = tid*k;

    // printf("%d\n", index);

    int orig_row = index / m;
    int orig_col = index % m;

    int last_row = n*(m+1);
    int row=orig_row, col=orig_col;

    if (orig_row < n) {
        for(int i=0; i<k; i++) {
            int oc_plus_i = orig_col+i;
            
            col = oc_plus_i%m;
            row = orig_row + (oc_plus_i/m);

            // printf("%d %d\n", row, col);

            int val = mat[row*(m+1)+col];
            atomicAdd(&mat[row*(m+1) + m], val);
            atomicAdd(&mat[last_row + col], val);
        }
    }
}

// Min value to be added to each element
__device__ int min_el = INT_MAX;

__global__ void findMin(int* mat, int n, int m, int k) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int orig_index = tid*k;
    int val = INT_MAX;
    int index = orig_index;

    for (int i=0; i<k; i++) {
        index = orig_index + i;

        if (index < n) {
            // Check in last col of each row
            val = mat[index*(m+1) + m];
        }
        else if (index < n+m) {
            // Check in last row
            val = mat[n*(m+1) + (index-n)];
        }
        else
            return;

        if (min_el > val)
            atomicMin(&min_el, val);
    }
}

__global__ void updateMin(int* mat, int rows, int cols, int k) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    int index = tid*k;

    if (index < rows*cols) {
        for(int i=0; i<k; i++) {
            mat[index+i]+=min_el;
        }
    }
}


int main() {
    int n,m,k;
    scanf("%d %d %d", &n, &m, &k);
    int *mat, *dmat;
    mat = (int*)calloc((n+1)*(m+1), sizeof(int));

    hipMalloc(&dmat, (n+1)*(m+1)*sizeof(int));


    for(int i=0; i<n; i++) {
        int row = i*(m+1);
        for(int j=0; j<m; j++) {
            scanf("%d", &mat[row+j]);
        }
    }

    // Initialize matrix
    // for(int i=0; i<n; i++) {
    //     for(int j=0; j<m; j++) {
    //         mat[i*(m+1) + j] = rand()%9 + 1;
    //     }
    // }


    hipMemcpy(dmat, mat, (n+1)*(m+1)*sizeof(int), hipMemcpyHostToDevice);
    int gridDim = ceil((float)(n*m) / (1024*k) );
    sumRandC<<<gridDim, 1024>>>(dmat, n, m, k);

    hipDeviceSynchronize();

    gridDim = ceil((float)(n+m)/(1024*k));
    findMin<<<gridDim, 1024>>>(dmat, n, m, k);

    hipDeviceSynchronize();

    gridDim = ceil((float)((n+1)*(m+1)) / (1024*k) );
    updateMin<<<gridDim, 1024>>>(dmat, n+1, m+1, k);

    hipMemcpy(mat, dmat, (n+1)*(m+1)*sizeof(int), hipMemcpyDeviceToHost);

    print_matrix(mat,n+1,m+1);
    return 0;
}
