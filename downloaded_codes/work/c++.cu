#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

class A {
public:
	__host__ __device__ A(unsigned ongpu = 1) { printf("in A's constructor: on %s.\n", (ongpu ? "GPU" : "CPU")); }
};
__global__ void dkernel(unsigned n) {
	A a;
	//printf("in dkernel %d\n", blockIdx.x * blockDim.x + threadIdx.x);
}

#define BLOCKSIZE	32
int main() {
	A b(0);
	unsigned N = BLOCKSIZE;
    	dkernel<<<1, BLOCKSIZE>>>(N);
	hipDeviceSynchronize();
    	return 0;
}
