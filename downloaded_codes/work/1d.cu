#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(unsigned *matrix) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	matrix[id] = id;
}
#define N	5
#define M	6
int main() {
	unsigned *matrix, *hmatrix;
	hipMalloc(&matrix, N * M * sizeof(unsigned));
	hmatrix = (unsigned *)malloc(N * M * sizeof(unsigned));
    	dkernel<<<N, M>>>(matrix);
	hipMemcpy(hmatrix, matrix, N * M * sizeof(unsigned), hipMemcpyDeviceToHost);
	for (unsigned ii = 0; ii < N; ++ii) {
		for (unsigned jj = 0; jj < M; ++jj) {
			printf("%2d ", hmatrix[ii * M + jj]);
		}
		printf("\n");
	}
    return 0;
}
