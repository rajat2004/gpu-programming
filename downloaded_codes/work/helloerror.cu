#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
const char *msg = "Hello World.\n";
__global__ void dkernel() {
    printf(msg);
}
int main() {
    dkernel<<<1, 32>>>();
	hipDeviceSynchronize();
    return 0;
}
