#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(char *arr, int arrlen) {
	unsigned id = threadIdx.x;
	if (id < arrlen) {
		++arr[id];
	}
}
int main() {
	char cpuarr[] = "Gdkkn\x1fVnqkc-", *gpuarr;

	hipMalloc(&gpuarr, sizeof(char) * (1 + strlen(cpuarr)));
	hipMemcpy(gpuarr, cpuarr, sizeof(char) * (1 + strlen(cpuarr)), hipMemcpyHostToDevice);
	dkernel<<<1, 32>>>(gpuarr, strlen(cpuarr));
	hipDeviceSynchronize();	// unnecessary.
	hipMemcpy(cpuarr, gpuarr, sizeof(char) * (1 + strlen(cpuarr)), hipMemcpyDeviceToHost);
	printf(cpuarr);

	return 0;
}
