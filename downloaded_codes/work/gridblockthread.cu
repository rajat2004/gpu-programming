#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(unsigned *nthreads) {
	if (threadIdx.x == 0 && blockIdx.x == 0 && threadIdx.y == 0 && blockIdx.y == 0 && threadIdx.z == 0 && blockIdx.z == 0) {
	//if (threadIdx.x == 0) {
    		printf("%d %d %d %d %d %d.\n", gridDim.x, gridDim.y, gridDim.z, blockDim.x, blockDim.y, blockDim.z);
	}
	atomicInc(nthreads, 1000000);
}
int main() {
	dim3 grid(2, 3, 4);
	dim3 block(5, 6, 7);
	unsigned *nthreads, hnthreads = 0;
	hipMalloc(&nthreads, sizeof(unsigned));
	hipMemcpy(nthreads, &hnthreads, sizeof(unsigned), hipMemcpyHostToDevice);
    dkernel<<<grid, block>>>(nthreads);
	hipMemcpy(&hnthreads, nthreads, sizeof(unsigned), hipMemcpyDeviceToHost);
	printf("%d\n", hnthreads);
    return 0;
}
