#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(unsigned *vector, unsigned vectorsize) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	switch(id) {
	case 0: vector[id] = 0; break;
	case 1: vector[id] = vector[id]; break;
	case 2: vector[id] = vector[id - 2]; break;
	case 3: vector[id] = vector[id + 3]; break;
	case 4: vector[id] = 4 + 4 + vector[id]; break;
	case 5: vector[id] = 5 - vector[id]; break;
	case 6: vector[id] = vector[6]; break;
	case 7: vector[id] = 7 + 7; break;
	case 8: vector[id] = vector[id] + 8; break;
	case 9: vector[id] = vector[id] * 9; break;
	}
}
#define BLOCKSIZE	10
#define N		BLOCKSIZE
int main(int nn, char *str[]) {
	unsigned *vector, *hvector;
	hipMalloc(&vector, N * sizeof(unsigned));
	hvector = (unsigned *)malloc(N * sizeof(unsigned));

	unsigned nblocks = ceil((float)N / BLOCKSIZE);

    	dkernel<<<nblocks, BLOCKSIZE>>>(vector, N);
	hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);
	for (unsigned ii = 0; ii < N; ++ii) {
		printf("%4d ", hvector[ii]);
	}
	printf("\n");
    	return 0;
}
