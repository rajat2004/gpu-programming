#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void K() {
	// Original: if (condition) atomicInc(&counter, 1000000);
	unsigned val = __ballot(threadIdx.x < 100);
	// leader.
	unsigned wcount = __popc(val);
	if (threadIdx.x % 32 == 0) printf("%d\n", __popc(val));
}
int main() {
	K<<<5, 128>>>();
	hipDeviceSynchronize();

	return 0;
}
