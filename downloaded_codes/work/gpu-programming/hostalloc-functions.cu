#include <hip/hip_runtime.h>
#include <stdio.h>

__host__ __device__ void fun(int *counter) {
	++*counter;
}
__global__ void printk(int *counter) {
	fun(counter);
	printf("printk (after fun): %d\n", *counter);
}
int main() {
	int *counter;

	hipHostAlloc(&counter, sizeof(int), 0);
	//cudaMalloc(&counter, sizeof(int));

	*counter = 0;
	printf("main: %d\n", *counter);

	printk <<<1, 1>>>(counter);
	hipDeviceSynchronize();

	fun(counter);
	printf("main (after fun): %d\n", *counter);

	return 0;
}
