#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printk(int *counter) {
	*counter <<= 1;
	printf("\t%d\n", *counter);
}
int main() {
	int hcounter = 1, *counter;

	hipMalloc(&counter, sizeof(int));
	do {
		printf("%d\n", hcounter);
		hipMemcpy(counter, &hcounter, sizeof(int), hipMemcpyHostToDevice);
		printk <<<1, 1>>>(counter);
		hipMemcpy(&hcounter, counter, sizeof(int), hipMemcpyDeviceToHost);
		hcounter <<= 1;
	} while (hcounter <= 100);
	return 0;
}
