
#include <hip/hip_runtime.h>
#include <stdio.h>


__global__ void onetoten() {
	__shared__ unsigned int n;
	n = 0;
	__syncthreads();

	while (n < 10) {
		int oldn = atomicInc(&n, 100);
		if (oldn % 3 == threadIdx.x) {
			printf("%d: %d\n", threadIdx.x, oldn);
		}
	}
}

__global__ void onetoten4() {
	__shared__ unsigned int n;
	n = 0;
	__syncthreads();

	while (n < 10) {
		int oldn = atomicInc(&n, 100);
		if (oldn % 3 == threadIdx.x) {
			printf("%d: %d\n", threadIdx.x, oldn);
		}
	}
}
__device__ volatile int n;
__global__ void onetoten3() {
	n = 0;
	__syncthreads();
	while (n < 10) {
		if (n % 3 == threadIdx.x) {
			printf("%d: %d\n", threadIdx.x, n);
			++n;
		}
	}
}
__global__ void onetoten2() {
	volatile __shared__ int n;
	n = 0;
	__syncthreads();
	while (n < 10) {
		if (n % 3 == threadIdx.x) {
			printf("%d: %d\n", threadIdx.x, n);
			++n;
		}
	}
}
__global__ void onetoten1() {
	__shared__ int n;
	n = 0;
	__syncthreads();
	while (n < 10) {
		if (n % 3 == threadIdx.x) {
			printf("%d: %d\n", threadIdx.x, n);
			++n;
		}
		__syncthreads();
	}
}
__global__ void onetoten0() {
	for (int ii = 0; ii < 10; ++ii) {
		if (ii % 3 == threadIdx.x) {
			printf("%d: %d\n", threadIdx.x, ii);
		}
	}
}
int main() {
	onetoten<<<1, 3>>>();
	hipDeviceSynchronize();
	return 0;
}
