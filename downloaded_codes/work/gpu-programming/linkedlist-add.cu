#include <stdio.h>
#include <hip/hip_runtime.h>

struct node {
	int data;
	struct node *next;
};

__device__ struct node *head;

__device__ struct node *getNewNode() {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	struct node *newnode = (struct node *)malloc(sizeof(struct node));
	newnode->data = id;
	newnode->next = NULL;
	return newnode;
}
__global__ void listAdd() {
	struct node *myoldhead, *actualoldhead;
	struct node *newnode = getNewNode();

	do {
		myoldhead = head;
		newnode->next = myoldhead;
		actualoldhead = (struct node *)atomicCAS((unsigned long long *)&head, (unsigned long long)myoldhead, (unsigned long long)newnode);
	} while (actualoldhead != myoldhead);
}
__device__ void listPrint(struct node *ptr) {
	printf("%d ", ptr->data);
}
__global__ void listPrint() {
	int nnodes = 0;
	for (struct node *ptr = head; ptr; ptr = ptr->next, ++nnodes)
		listPrint(ptr);
	printf("\nNumber of nodes = %d\n", nnodes);
}
int main() {
	hipMemset(&head, 0, sizeof(struct node *));	
	listAdd<<<4, 1024>>>();
	listPrint<<<1, 1>>>();
	hipDeviceSynchronize();

	return 0;
}
