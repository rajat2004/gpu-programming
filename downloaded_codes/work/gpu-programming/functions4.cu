#include <stdio.h>
#include <hip/hip_runtime.h>

#define N	10

__host__ __device__ void fun(int *arr, int ii) {
	++arr[ii];
}
__global__ void dfun(int *arr) {
	fun(arr, threadIdx.x);
}
__host__ __device__ void print(int *arr, int ii) {
	printf("%d, ", arr[ii]);
}
__global__ void dprint(int *arr) {
	print(arr, threadIdx.x);
}
int main() {
	int arr[N], *darr;

	hipMalloc(&darr, N * sizeof(int));

	for (unsigned ii = 0; ii < N; ++ii)
		arr[ii] = ii;
	hipMemcpy(darr, arr, N * sizeof(int), hipMemcpyHostToDevice);

	for (unsigned ii = 0; ii < N; ++ii)
		fun(arr, ii);
	dfun<<<1, N>>>(darr);
	hipDeviceSynchronize();

	for (unsigned ii = 0; ii < N; ++ii)
		print(arr, ii);
	printf("\n");
	dprint<<<1, N>>>(darr);
	hipDeviceSynchronize();
	printf("\n");

	return 0;
}
