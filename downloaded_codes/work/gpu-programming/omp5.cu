#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#define N 12

int sum = 0;
int main() {
	int *a = (int *)malloc(sizeof(int) * N);

	#pragma omp parallel for reduction(+:sum)
	for (int ii = 0; ii < N; ++ii) {
		a[ii] = ii + 1;
		sum += a[ii];
	}

	printf("sum = %d\n", sum);
	return 0;
}
