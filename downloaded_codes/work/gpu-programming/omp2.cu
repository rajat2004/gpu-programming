#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime.h>

__global__ void K() {
	printf("in K: %d\n", threadIdx.x);
}
// Compiler as: nvcc -Xcompiler -fopenmp -lgomp omp.cu
int main() {
	int sh = 5;
	#pragma omp parallel for
	for (int i = 0; i < 10; ++i)
	{
		K<<<1, 1>>>();
		hipDeviceSynchronize();
		++sh;
		printf("sh = %d\n", sh);
	}

	return 0;
}
