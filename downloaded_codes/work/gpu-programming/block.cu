#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#define K 32
#define N 32
__global__ void fun(int *a) {
	int i;
	unsigned nthreads = blockDim.x * gridDim.x;
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned start = N / nthreads * id;
	for (i = 0; i < N/K; ++i)
		a[start + i] = threadIdx.x * threadIdx.x;
}
double rtclock() {
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday(&Tp, &Tzp);
  if (stat != 0) printf("Error return from gettimeofday: %d", stat);
  return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}

void printtime(const char *str, double starttime, double endtime) {
	printf("%s%3f seconds\n", str, endtime - starttime);
}
int main() {
	int a[N], *da;
	int i;

	hipMalloc(&da, N * sizeof(int));
	double start = rtclock();
	for (i = 0; i < 1000; ++i) {
		fun<<<K, N>>>(da);
		hipDeviceSynchronize();
	}
	double end = rtclock();
	printtime("Single block: ", start, end);
	hipMemcpy(a, da, N * sizeof(int), hipMemcpyDeviceToHost);
	//for (i = 0; i < N; ++i)
	//	printf("%d\n", a[i]);
	return 0;
}
