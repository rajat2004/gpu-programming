#include <stdio.h>
#include <hip/hip_runtime.h>

#define N	100000

__device__ unsigned wlsize;
__device__ int worklist[N];

__global__ void k1() {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	worklist[atomicInc(&wlsize, N)] = id;
}
__global__ void k2() {
	printf("Number of elements added = %d\n", wlsize);
}
int main() {
	hipMemset(&wlsize, 0, sizeof(int));	// initialization.
	k1<<<4, 64>>>();
	hipDeviceSynchronize();
	k2<<<1, 1>>>();
	hipDeviceSynchronize();

	return 0;
}
