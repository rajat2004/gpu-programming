#include<stdio.h>
#include<hip/hip_runtime.h>

#define N		1024
#define BLOCKSIZE	64

__device__ unsigned binary[N];
__device__ volatile unsigned k2counter;		// try removing volatile: the code may hang.

__global__ void K() {
	unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
	binary[id] = id;
	__syncthreads();
	if (binary[N-1 - id] != N-1 - id)
		printf("Error: There is no global barrier.\n");
}
__global__ void K2init() {
	k2counter = 0;
}
__global__ void K2() {
	unsigned id = blockDim.x * blockIdx.x + threadIdx.x;

	printf("This is before: %d\n", id);

	// global barrier start
	atomicInc((unsigned *)&k2counter, N + 1);

	while (k2counter != N)
		;
	// global barrier end

	printf("This is after the global barrier: %d\n", id);
}
int main() {
	K<<<N / BLOCKSIZE, BLOCKSIZE>>>();

	K2init<<<1, 1>>>();
	K2<<<N / BLOCKSIZE, BLOCKSIZE>>>();
	hipDeviceSynchronize();

	return 0;
}
