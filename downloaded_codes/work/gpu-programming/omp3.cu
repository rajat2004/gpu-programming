
#include <hip/hip_runtime.h>
#include <omp.h>
#include <stdio.h>

int main(int argc, char *argv[]) {

 int nthreads, tid;

 /* Fork a team of threads with each thread having a private tid variable */
 #pragma omp parallel shared(tid)
   {

   /* Obtain and print thread id */
   tid = omp_get_thread_num();
   printf("%d\n", tid);

   /* Only master thread does this */
   if (tid == 0) 
     {
     nthreads = omp_get_num_threads();
     printf("Number of threads = %d\n", nthreads);
     }

   }  /* All threads join master thread and terminate */
 }

