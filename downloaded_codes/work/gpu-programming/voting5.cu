#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 64

__global__ void init(int *a) {
	a[threadIdx.x] = 1;
}
__global__ void K(int *a) {
	// this forces other threads to return false. Ideally, other threads should be don't care.
	//unsigned mask = __ballot(threadIdx.x % 3 == 0 && a[threadIdx.x] == 0);
	unsigned mask = __ballot(threadIdx.x % 3 == 0 && a[threadIdx.x] == 0 || threadIdx.x % 3 != 0);
	if (threadIdx.x % 32 == 0) {
		printf("%X\n", mask);
	}
}
int main() {
	int *a;
	hipMalloc(&a, N * sizeof(int));
	init<<<1, N>>>(a);
	K<<<1, N>>>(a);
	hipDeviceSynchronize();

	return 0;
}
