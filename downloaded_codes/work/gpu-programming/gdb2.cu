#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void K(int *x) {
	*x = 0;
	printf("%d\n", *x);
}
int main() {
	int *x = NULL;
	K<<<2, 10>>>(x);
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	printf("error=%d, %s, %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
	return 0;
}
