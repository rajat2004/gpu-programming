#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void K1() {
	unsigned num = 0;
	for (unsigned ii = 0; ii < threadIdx.x; ++ii)
		num += ii;
	printf("K1: %d\n", threadIdx.x);
}
__global__ void K2() {
	printf("K2\n");
}
int main() {
	hipStream_t s1, s2;
	hipStreamCreate(&s1);
	hipStreamCreate(&s2);

	K1<<<1, 1024, 0, s1>>>();
	K2<<<1, 32, 0, s2>>>();
	hipDeviceSynchronize();

	return 0;
}
