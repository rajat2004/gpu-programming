#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void K(int *x) {
	*x = 0;
	printf("%d\n", *x);
}
int main() {
	int *x = NULL;
	K<<<2, 10>>>(x);
	hipDeviceSynchronize();

	return 0;
}
