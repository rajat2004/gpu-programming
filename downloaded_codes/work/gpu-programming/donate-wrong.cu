#include <stdio.h>
#include <hip/hip_runtime.h>

#define N		500
#define BLOCKSIZE	64
#define ELEPERTHREAD	20


__device__ const unsigned delta = ELEPERTHREAD / 5;

__global__ void k1(unsigned *nelements) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	__shared__ unsigned sum;
	__shared__ unsigned avg;
	__shared__ unsigned donationbox[N], donationboxindex;

	if (id == 0) {
		sum = 0;
		donationboxindex = 0;
	}
	
	// compute sum.
	atomicAdd(&sum, nelements[id]);

	// compute average.
	if (id == 0) avg = sum / blockDim.x;

	// check if I need to donate.
	unsigned surplus = nelements[id] - avg;
	if (surplus > delta) {
		// donate.
		unsigned index = atomicAdd(&donationboxindex, surplus);
		for (unsigned ii = 0; ii < surplus; ++ii) {
			donationbox[index + ii] = id;	// some work.
		}
	}

	// process.
	// some processing here.
	__syncthreads();

	// empty donation box.
	while (donationboxindex < N * ELEPERTHREAD) {
		unsigned index = atomicDec(&donationboxindex, N * ELEPERTHREAD + blockDim.x);	// to ensure that wrap-around does not cause confusion.
		if (index < N * ELEPERTHREAD) {
			unsigned work = donationbox[index];
			// process with work.
		}
	}
}

int main() {
	unsigned hnelements[N];
	for (unsigned ii = 0; ii < N; ++ii) {
		hnelements[ii] = rand() % ELEPERTHREAD;
	}

	unsigned *nelements;
	hipMalloc(&nelements, N * sizeof(unsigned));
	hipMemcpy(nelements, hnelements, N * sizeof(unsigned), hipMemcpyHostToDevice);

	unsigned nblocks = (N + BLOCKSIZE - 1) / BLOCKSIZE;
	k1<<<nblocks, BLOCKSIZE>>>(nelements);
	hipDeviceSynchronize();
	//k2<<<1, 1>>>();
	//cudaDeviceSynchronize();

	return 0;
}
