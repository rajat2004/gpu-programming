#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void K1() {
	printf("in K1\n");
}
__global__ void K2() {
	printf("in K2\n");
}
__global__ void K3() {
	printf("in K3\n");
}
__global__ void K4() {
	printf("in K4\n");
}
int main() {
	hipStream_t s0, s1;
	hipEvent_t e0, e1;

	hipSetDevice(0);
	hipStreamCreate(&s0);
	hipEventCreate(&e0);

	K1<<<1, 1, 0, s0>>>();
	hipEventRecord(e0, s0);
	K2<<<1, 1, 0, s0>>>();

	hipSetDevice(1);
	hipStreamCreate(&s1);
	hipEventCreate(&e1);

	K3<<<1, 1, 0, s1>>>();
	hipStreamWaitEvent(s1, e0, 0);
	K4<<<1, 1, 0, s1>>>();

	hipDeviceSynchronize();

	hipSetDevice(0);
	hipDeviceSynchronize();
	return 0;
}
