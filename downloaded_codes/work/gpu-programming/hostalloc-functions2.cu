#include <hip/hip_runtime.h>
#include <stdio.h>

__device__ int counter;
__host__ __device__ void fun() {
	++counter;
}
__global__ void printk() {
	fun();
	printf("printk (after fun): %d\n", counter);
}
int main() {

	//counter = 0;
	//printf("main: %d\n", counter);

	printk <<<1, 1>>>();
	hipDeviceSynchronize();

	//fun();
	//printf("main (after fun): %d\n", counter);

	return 0;
}
