#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE	26

__global__ void dkernel() {
	__shared__ char str[BLOCKSIZE+1];
	str[threadIdx.x] = 'A' + (threadIdx.x + blockIdx.x) % BLOCKSIZE;
	if (threadIdx.x == 0) {
		str[BLOCKSIZE] = '\0';
	}
	//__syncthreads();
	if (threadIdx.x == 0) {
		printf("%d: %s\n", blockIdx.x, str);
	}
}
int main() {
	dkernel<<<10, BLOCKSIZE>>>();
	hipDeviceSynchronize();
}
