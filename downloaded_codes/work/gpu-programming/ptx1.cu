#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void K() {
	printf("in K\n");
}
int main() {
	K<<<1, 1>>>();
	hipDeviceSynchronize();

	return 0;
}
