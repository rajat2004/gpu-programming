#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

texture<float, 2, hipReadModeElementType> tex;

__global__ void transformKernel(float *output, int width, int height, float theta) {
	unsigned x = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned y = blockIdx.y * blockDim.y + threadIdx.y;

	float u = (float)x - (float)width / 2;
	float v = (float)y - (float)height / 2;
	float tu = (u * cosf(theta) - v * sinf(theta)) / width;
	float tv = (v * cosf(theta) + u * sinf(theta)) / height;

	output[y * width + x] = tex2D(tex, tu + 0.5, tv + 0.5);
}
int main() {
	int width = 5, height = 5;
	unsigned size = width * height * sizeof(float);
	float *hData = (float *)malloc(size);
	for (unsigned ii = 0; ii < width; ++ii)
		for (unsigned jj = 0; jj < height; ++jj)
			hData[ii * height + jj] = ii + jj;
    	hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 0, 0, 0, hipChannelFormatKindFloat);
	hipArray *cuArray;
	hipMallocArray(&cuArray, &channelDesc, width, height);
	hipMemcpyToArray(cuArray, 0, 0, hData, size, hipMemcpyHostToDevice);

	tex.addressMode[0] = hipAddressModeWrap;
	tex.addressMode[1] = hipAddressModeWrap;
	tex.filterMode     = hipFilterModeLinear;
	tex.normalized     = true;
	hipBindTextureToArray(tex, cuArray, channelDesc);

	float *dData;
	hipMalloc(&dData, size);
	dim3 block(8, 8, 1);
	dim3 grid(width / block.x, height / block.y, 1);;
	transformKernel<<<grid, block>>>(dData, width, height, 0.6);
	return 0;
}
