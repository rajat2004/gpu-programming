#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void K() {
	//unsigned val = __ballot(threadIdx.x % 2 == 0);
	if (threadIdx.x % 2 == 0) {
		unsigned val = __ballot(threadIdx.x < 100);
		if (threadIdx.x % 32 == 0) printf("%d\n", __popc(val));
	}
}
int main() {
	K<<<1, 128>>>();
	hipDeviceSynchronize();

	return 0;
}
