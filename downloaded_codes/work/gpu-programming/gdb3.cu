#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void K(int *p) {
	*p = 0;
	printf("%d\n", *p);
}
int main() {
	int *x, *y;
	hipMalloc(&x, sizeof(int));

	K<<<2, 10>>>(x);
	hipDeviceSynchronize();

	y = x;
	hipFree(y);

	K<<<2, 10>>>(x);
	hipDeviceSynchronize();
	//cudaError_t err = cudaGetLastError();
	//printf("error=%d, %s, %s\n", err, cudaGetErrorName(err), cudaGetErrorString(err));

	return 0;
}
