#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#define N 100
__global__ void K(int *a, int start, int end) {
	printf("start = %d, end = %d\n", start, end);
}
int main() {
	int a[N];
	int ii;

	omp_set_num_threads(5);
	#pragma omp parallel
	{
	#pragma omp parallel for
	for (ii = 0; ii < N; ++ii) {
		a[ii] = ii;
	}

	int nthreads = omp_get_num_threads();
	int perthread = N / nthreads;
	int start = perthread * omp_get_thread_num();
	int end = start + perthread;
	K<<<1, 1>>>(a, start, end);
	hipDeviceSynchronize();
	
	}
	printf("All over.\n");

	return 0;
}
