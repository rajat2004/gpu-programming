#include <stdio.h>
#include <hip/hip_runtime.h>


#define N		100
#define	BLOCKSIZE	32

__global__ void init(int *input) {
	unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id < N) input[id] = id + 1;
}
__global__ void print(int *output) {
	for (unsigned ii = 0; ii < N; ++ii)
		printf("%d ", output[ii]);
	printf("\n");
}
__global__ void convolution(int *input, int *filter, int *output, int fsize) {
	unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
	if (id >= N) return;

	//int *filteroutput = (int *)malloc(fsize * sizeof(int));
	int sum = 0;
	int halff = fsize / 2;
	int istart = id - halff, iend = id + halff + 1;
	int fstart = 0, fend = fsize;

	if (istart < 0) {
		fstart -= istart;
		istart = 0;
	}
	if (iend > N) {
		fend -= (iend - N);
		iend = N;
	}
	
	for (unsigned ii = fstart; ii < fend; ++ii) {
		// filteroutput[ii] = input[id + ii] * filter[ii];
		sum += input[istart + ii - fstart] * filter[ii];
	}
	output[id] = sum;
}
int main() {
	int *input, *filter, *output;
	int hf[] = {3, 4, 5, 4, 3};
	int fsize = sizeof(hf) / sizeof(*hf);

	if (fsize % 2 == 0) {
		printf("Error: Filter size (%d) is even.\n", fsize);
		exit(1);
	}
	hipMalloc(&input, N * sizeof(int));
	hipMalloc(&filter, fsize * sizeof(int));
	hipMalloc(&output, N * sizeof(int));

	hipMemcpy(filter, hf, fsize * sizeof(int), hipMemcpyHostToDevice);
	
	int nblocks = (N + BLOCKSIZE - 1) / BLOCKSIZE;
	init<<<nblocks, BLOCKSIZE>>>(input);

	convolution<<<nblocks, BLOCKSIZE>>>(input, filter, output, fsize);

	print<<<1, 1>>>(output);
	hipDeviceSynchronize();

	return 0;
}
