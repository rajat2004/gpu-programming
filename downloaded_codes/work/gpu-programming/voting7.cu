#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void K() {
	int x = threadIdx.x;
	unsigned mask = __match_any_sync(x);
	if (threadIdx.x % 32 == 0) printf("%X\n", mask);
}
int main() {
	K<<<1, 128>>>();
	hipDeviceSynchronize();

	return 0;
}
