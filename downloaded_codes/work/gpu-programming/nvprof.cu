#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void K1(int num) {
	num += num;
	++num;
}
__device__ int sum = 0;
__global__ void K2(int num) {
	atomicAdd(&sum, num);
}
__global__ void K3(int num) {
	__shared__ int sum;
	sum = 0;
	__syncthreads();

	sum += num;
}
int main() {
	for (unsigned ii = 0; ii < 100; ++ii) {
		K1<<<5, 32>>>(ii);
		hipDeviceSynchronize();
	}
	for (unsigned ii = 0; ii < 100; ++ii) {
		K2<<<5, 32>>>(ii);
		hipDeviceSynchronize();
	}
	for (unsigned ii = 0; ii < 100; ++ii) {
		K3<<<5, 32>>>(ii);
		hipDeviceSynchronize();
	}
	return 0;
}
