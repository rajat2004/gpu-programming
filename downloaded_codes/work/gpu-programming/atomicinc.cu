#include <stdio.h>
#include <hip/hip_runtime.h>
#define SIZE 64

#define cudaCheckError() {                                          \
 hipError_t e=hipGetLastError();                                 \
 if(e!=hipSuccess) {                                              \
   printf("Cuda failure %s:%d: '%s'\n",__FILE__,__LINE__,hipGetErrorString(e)); \
   exit(0); \
 }                                                                 \
}

__global__ void kernel(unsigned int * x){
  atomicInc(&x[0],100);
  __syncthreads();
  printf("hi I'm tid %u - %u\n", threadIdx.x, x[0]);
  
}
int main(){
  //! unsigned int* x;
  //! x = (unsigned int*) malloc(sizeof(unsigned int) * SIZE);
  //! *x=0;
  
  unsigned int* dx;
  hipMalloc( (void**) &dx, SIZE*sizeof(unsigned int));  cudaCheckError();
  
  //! cudaMemcpy(dx ,x , sizeof(unsigned int)* SIZE , cudaMemcpyHostToDevice); // did not make a difference
  
  kernel<<< 1, SIZE >>>(dx);   
  hipDeviceSynchronize();  cudaCheckError();
  return 0;
}
