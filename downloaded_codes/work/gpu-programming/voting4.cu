#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void K() {
	// Original: if (condition) atomicInc(&counter, 1000000);
	unsigned mask = __ballot(condition);
	if (threadIdx.x % 32 == 0) {
		atomicAdd(&counter, __popc(mask));
	}
	// leader.
	//unsigned wcount = __popc(val);
	//if (threadIdx.x % 32 == 0) printf("%d\n", __popc(val));
}
int main() {
	K<<<5, 64>>>();
	hipDeviceSynchronize();

	return 0;
}
