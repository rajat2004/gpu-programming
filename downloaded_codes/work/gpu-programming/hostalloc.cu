#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printk(int *counter) {
	++*counter;
	printf("\t%d\n", *counter);
}
int main() {
	int *counter;

	hipHostAlloc(&counter, sizeof(int), 0);
	//cudaHostAlloc(&counter, sizeof(int), cudaHostAllocMapped);
	do {
		printf("%d\n", *counter);
		printk <<<1, 1>>>(counter);
		hipDeviceSynchronize();
		++*counter;
	} while (*counter < 10);

	hipHostFree(counter);
	return 0;
}
