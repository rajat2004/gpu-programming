#include <stdio.h>
#include <hip/hip_runtime.h>

#define N		1024		// must be a power of 2.
#define BLOCKSIZE	N


__global__ void RKPlusNBy2(unsigned *nelements) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	for (int off = N / 2; off; off /= 2) {
		if (id < off)
			nelements[id] += nelements[id + off];
		__syncthreads();
	}
	if (id == 0)
		printf("GPU sum = %d\n", *nelements);
}
__global__ void RKNminusI(unsigned *nelements) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	for (int off = N / 2; off; off /= 2) {
		if (id < off)
			nelements[id] += nelements[2 * off - id - 1];
		__syncthreads();
	}
	if (id == 0)
		printf("GPU sum = %d\n", *nelements);
}
__global__ void RKConsecutive(unsigned *nelements) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	for (int off = N / 2; off; off /= 2) {
		if (id < off)
			nelements[N / off * id] += nelements[N / off * id + N / 2 / off];
		__syncthreads();
	}
	if (id == 0)
		printf("GPU sum = %d\n", *nelements);
}

int main() {
	unsigned hnelements[N];
	unsigned sum = 0;
	for (unsigned ii = 0; ii < N; ++ii) {
		hnelements[ii] = rand() % 20;
		sum += hnelements[ii];
	}
	printf("CPU sum = %d\n", sum);

	unsigned nblocks = (N + BLOCKSIZE - 1) / BLOCKSIZE;

	unsigned *nelements;
	hipMalloc(&nelements, N * sizeof(unsigned));

	hipMemcpy(nelements, hnelements, N * sizeof(unsigned), hipMemcpyHostToDevice);
	RKPlusNBy2<<<nblocks, BLOCKSIZE>>>(nelements);

	hipMemcpy(nelements, hnelements, N * sizeof(unsigned), hipMemcpyHostToDevice);
	RKNminusI<<<nblocks, BLOCKSIZE>>>(nelements);

	hipMemcpy(nelements, hnelements, N * sizeof(unsigned), hipMemcpyHostToDevice);
	RKConsecutive<<<nblocks, BLOCKSIZE>>>(nelements);

	hipDeviceSynchronize();

	return 0;
}
