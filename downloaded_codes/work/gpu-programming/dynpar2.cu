#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define K 2

__global__ void Child(int father) {
	printf("%d\n", father + threadIdx.x);
}
__global__ void Parent() {
	if (threadIdx.x % K == 0) {
		Child<<<1, K>>>(threadIdx.x);
		hipDeviceSynchronize();
		printf("Called childen with starting %d\n", threadIdx.x);
	}
}
int main() {
	Parent<<<1, 10>>>();
	hipDeviceSynchronize();

	return 0;
}
