#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "mytime.h"

#define N 1024
__global__ void dkernel(unsigned *a, unsigned wpt, unsigned chunksize) {
	for (unsigned ii = 0; ii < wpt; ii += chunksize) {
		unsigned start = wpt * blockDim.x * threadIdx.x;
		for (unsigned nn = start; nn < start + chunksize; ++nn) {
			a[nn]++;
		}
	}
}
int main() {
	unsigned *a;
	double start, end;
	int i;

	hipMalloc(&a, sizeof(unsigned) * N);

	for (i = 1; i < 33; ++i) {
		start = rtclock();
		dkernel<<<1, 32>>>(a, N / 32, i);
		hipDeviceSynchronize();
		end = rtclock();
		printf("%3d: ", i);
		printtime("", start, end);
	}
    return 0;
}
