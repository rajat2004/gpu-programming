#include <hip/hip_runtime.h>
#include <stdio.h>

int main() {
	hipEvent_t start, stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);

	hipEventRecord(start, 0);
	printf("Hello World\n");
	unsigned sum = 0;
	for (unsigned ii = 0; ii < 100000; ++ii)
		sum += ii;
	hipEventRecord(stop, 0);

	float elapsedtime;
	hipEventElapsedTime(&elapsedtime, start, stop);
	printf("time = %f ms\n", elapsedtime);

	return 0;
}
