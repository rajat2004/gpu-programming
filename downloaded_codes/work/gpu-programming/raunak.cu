#include <stdio.h>
#include <hip/hip_runtime.h>
__device__ volatile int counter = 0;
//Launching kernel with <<< (4,4) , (7,7) >>> gridDim.x =4 gridDim.y =4
__global__ void stencil(int* d_input, int M, int N)
{
// TODO: Your implementation goes here
        int tot_blocks = gridDim.x*gridDim.y; // =16
        int thidx = blockDim.x*blockIdx.x + threadIdx.x;
        int thidy = blockDim.y*blockIdx.y + threadIdx.y;
        int write_this;
        if (thidx>0 && thidy >0 && thidx <N-1 && thidy < M-1)
                write_this = 0.2*(d_input[thidy*N+thidx] + d_input[(thidy+1)*N+thidx] + d_input[(thidy-1)*N+thidx] + d_input[thidy*N+thidx+1] + d_input[thidy*N+thidx-
1]);
        __syncthreads();
        if(threadIdx.x==0 && threadIdx.y==0)
                atomicAdd((int *)&counter,1);
        while(counter<tot_blocks); // Waits here for infinite time.
        if (thidx>0 && thidy >0 && thidx <N-1 && thidy < M-1)
                d_input[thidy*N+thidx] = write_this;
}

int main() {
	int *arr;
	const int M = 16, N = 16;

	hipMalloc(&arr, M * N * sizeof(int));
	stencil<<<M, N>>>(arr, M, N);
	hipDeviceSynchronize();
}
