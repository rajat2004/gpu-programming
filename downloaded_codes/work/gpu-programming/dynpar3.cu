#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ int value;
__global__ void child() {
	printf("in child %d\n", threadIdx.x);
}
__device__ void dchild() {
	child<<<1, 10>>>();
	hipDeviceSynchronize();
}
__global__ void parent() {
	dchild();
}
int main() {
	parent<<<1, 2>>>();
	hipDeviceSynchronize();
}
