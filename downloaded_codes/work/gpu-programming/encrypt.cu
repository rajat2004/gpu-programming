#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdlib.h>

#define BLOCKSIZE	1024

__device__ char dec(char c) {
	return (c - 1);
}
__device__ char enc(char c) {
	return (c + 1);
}
__global__ void decrypt(char *src, char *target, int n) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n)
		target[id] = dec(src[id]);
	else if (id == n)
		target[id] = '\0';
}
__global__ void encrypt(char *src, char *target, int n) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n)
		target[id] = enc(src[id]);
	else if (id == n)
		target[id] = '\0';
}
void init(char *s, int *n) {
	strcpy(s, "Hello World!");
	*n = strlen(s);
}
int main() {
	char *s, *ds;

	int n;
	s = (char *)malloc(20);
	init(s, &n);

	dim3 block(BLOCKSIZE, 1, 1);
	dim3 grid(ceil((float)n/BLOCKSIZE), 1, 1);
	printf("number of blocks = %d\n", ceil((float)n/BLOCKSIZE));


	hipMalloc(&ds, (n + 1)*sizeof(char));

	hipMemcpy(ds, s, (n+1)*sizeof(char), hipMemcpyHostToDevice);
	encrypt<<<grid, block>>>(ds, ds, n);
	hipDeviceSynchronize();

	hipMemcpy(s, ds, (n+1)*sizeof(char), hipMemcpyDeviceToHost);
	puts(s);

	decrypt<<<grid, block>>>(ds, ds, n);
	hipDeviceSynchronize();

	hipMemcpy(s, ds, (n+1)*sizeof(char), hipMemcpyDeviceToHost);
	puts(s);

	printf("\n");
	return 0;
}
