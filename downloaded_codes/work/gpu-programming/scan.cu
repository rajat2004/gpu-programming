#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N	64
__global__ void exscan() {
	__shared__ unsigned a[N]; //= {4, 3, 9, 3, 5, 7, 3, 2};
	a[threadIdx.x] = threadIdx.x;
	__syncthreads();

	unsigned n = sizeof(a) / sizeof (*a);
	__syncthreads();

	if (threadIdx.x == 0) {
		for (unsigned ii = 0; ii < n; ++ii)
			printf("%d ", a[ii]);
		printf("\n");
	}
	__syncthreads();
	
	int tmp;
	for (int off = 1; off < n; off *= 2) {
		if (threadIdx.x >= off) {
			tmp = a[threadIdx.x - off];
		}
		__syncthreads();
		if (threadIdx.x >= off) {
			a[threadIdx.x] += tmp;
		}
		__syncthreads();
	}
	if (threadIdx.x == 0) {
		for (unsigned ii = 0; ii < n; ++ii)
			printf("%d ", a[ii]);
		printf("\n");
	}
}

int main() {
	//hipSetDevice(5);
	exscan<<<1, N>>>();
	hipDeviceSynchronize();
	return 0;
}

