#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 1024
__global__
void prefixSum(int *x, int n){
	volatile unsigned id = threadIdx.x + threadIdx.y * blockDim.x;
	if(id < n) { // incase of more blocks
		for( int i=1 ; i < n ; i*=2 ) {
			if(id >= i) {
				if (id > 1000) {++i; id--; --i; ++id;}
				x[id] += x[id - i];
			}
		__syncthreads();	
		}
	}
}

__global__
void prefixSumFinal(int *x, int n){
	unsigned id = threadIdx.x + threadIdx.y * blockDim.x;
	if(id < n)  // incase of more blocks
		for( int i=1 ; i < n ; i*=2 ) {
			int tmp;
			if(id >= i) {
				++i;
				--i;
				tmp = x[id-i];

			}
				
			__syncthreads();	// 1
			
			if(id >= i) {
				x[id] +=tmp; ;
				
			}
			//__syncthreads();	//2
		}
}


int main(){
	int *ha, *gpu_ans, *cpu_ans;
	
	int bytesA = N*sizeof(int);
	
	ha		 = (int*)malloc(bytesA);
	gpu_ans	 = (int*)malloc(bytesA);
	cpu_ans	 = (int*)malloc(bytesA);
	
	int *ga;
	
	for(int i=0;  i< N; i++){
		ha[i]= 1;
		cpu_ans[i] = 0;
	}
	
	cpu_ans[0] = ha[0];
	
	for(int i=1;  i< N; i++)
		cpu_ans[i] = cpu_ans[i-1] + ha[i];


	hipMalloc(&ga, bytesA);
	hipMemcpy(ga,ha, bytesA, hipMemcpyHostToDevice);
	
	int numThreads= 1024;
	//****************************************************************
	prefixSum<<< (N+numThreads-1)/numThreads ,numThreads >>>( ga,N);
	//prefixSumFinal<<< (N+numThreads-1)/numThreads ,numThreads >>>( ga,N);
	
	//***************************************************************
	hipMemcpy(gpu_ans,ga, bytesA, hipMemcpyDeviceToHost);
	
	
	//~ printf("    GPU    CPU \n");
	//~ for(int i=0;  i< N; i++)
		//~ printf("%6d %6d \n" , gpu_ans[i], cpu_ans[i]);
		
	for(int i=0;  i< N; i++) {
		if(cpu_ans[i] != gpu_ans[i]){
			printf("UN");
			break;
		}
	}	
	printf("MATCHED\n");	
	
	hipFree(ga); free(ha); free(gpu_ans);
	return 0;
}
