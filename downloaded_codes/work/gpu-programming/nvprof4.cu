#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ int sumg = 0;
__global__ void K() {
	int num = blockIdx.x * blockDim.x + threadIdx.x;
	num += num;
	++num;
	atomicAdd(&sumg, num);
	__shared__ int sum;
	sum = 0;
	__syncthreads();

	sum += num;
}
int main() {
	K<<<100, 32*5>>>();
	hipDeviceSynchronize();
	return 0;
}
