#include <stdio.h>
#include <hip/hip_runtime.h>


__global__ void MyKernel() {
	printf("in mykernel\n");
}
void MyCallback(hipStream_t stream, hipError_t status, void *data){
    printf("Inside callback %d\n", (long)data);
	MyKernel<<<1, 1>>>();
	hipDeviceSynchronize();
	hipError_t err = hipGetLastError();
	printf("error=%d, %s, %s\n", err, hipGetErrorName(err), hipGetErrorString(err));
}
int main() {
hipStream_t stream[2];
for (long i = 0; i < 2; ++i) {
	hipStreamCreate(&stream[i]);
    //cudaMemcpyAsync(devPtrIn[i], hostPtr[i], size, cudaMemcpyHostToDevice, stream[i]);
    MyKernel<<<1, 1, 0, stream[i]>>>();
    //cudaMemcpyAsync(hostPtr[i], devPtrOut[i], size, cudaMemcpyDeviceToHost, stream[i]);
    hipStreamAddCallback(stream[i], MyCallback, (void*)i, 0);
    MyKernel<<<1, 1, 0, stream[i]>>>();
	hipDeviceSynchronize();
}
}
