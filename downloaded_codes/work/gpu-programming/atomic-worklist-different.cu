#include <stdio.h>
#include <hip/hip_runtime.h>

#define N		500
#define BLOCKSIZE	64
#define ELEPERTHREAD	5

__device__ unsigned wlsize;
__device__ unsigned worklist[N * ELEPERTHREAD];

__global__ void k1(unsigned *nelements) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	unsigned index = atomicAdd(&wlsize, nelements[id]);
	for (unsigned ii = 0; ii < nelements[id]; ++ii)
		worklist[index + ii] = id;
}
__global__ void k2() {
	printf("Number of threads = %d, worklist size = %d\n", N, wlsize);

	for (unsigned ii = 0; ii < wlsize; ++ii)
		printf("%d ", worklist[ii]);
	printf("\n");
}
int main() {
	hipMemset(&wlsize, 0, sizeof(unsigned));	// initialization.

	unsigned hnelements[N];
	for (unsigned ii = 0; ii < N; ++ii) {
		hnelements[ii] = rand() % ELEPERTHREAD;
	}

	unsigned *nelements;
	hipMalloc(&nelements, N * sizeof(unsigned));
	hipMemcpy(nelements, hnelements, N * sizeof(unsigned), hipMemcpyHostToDevice);

	unsigned nblocks = (N + BLOCKSIZE - 1) / BLOCKSIZE;
	k1<<<nblocks, BLOCKSIZE>>>(nelements);
	hipDeviceSynchronize();
	k2<<<1, 1>>>();
	hipDeviceSynchronize();

	return 0;
}
