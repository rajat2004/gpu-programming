#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ int sumg = 0;
__global__ void K(int num) {
	num += num;
	++num;
	atomicAdd(&sumg, num);
	__shared__ int sum;
	sum = 0;
	__syncthreads();

	sum += num;
}
int main() {
	for (unsigned ii = 0; ii < 100; ++ii) {
		K<<<5, 32>>>(ii);
		hipDeviceSynchronize();
	}
	return 0;
}
