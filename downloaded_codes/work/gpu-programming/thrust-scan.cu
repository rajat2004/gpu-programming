#include "hip/hip_runtime.h"
#include <thrust/scan.h>
#include <thrust/functional.h>

/*
	Output: 1 1 1 2 2 2 4 4 4 4
	Explanation:
	init = 		1
	1 max -5 = 	1
	1 max 0  =	1
	1 max 2  =	2
	2 max -3 = 	2
	2 max 2  = 	2
	2 max 4  =	4
	4 max 0  = 	4
	4 max -1 = 	4
	4 max 2  =	4
	NOT performed: 4 max 8 = 8, as this is exclusive scan.
*/
int main() {
	int data[] = {-5, 0, 2, -3, 2, 4, 0, -1, 2, 8};
	int sizedata = sizeof(data) / sizeof(*data);
	thrust::maximum<int> binop;
	thrust::exclusive_scan(data, data + sizedata, data, 1, binop);
	//thrust::inclusive_scan(data, data + sizedata, data, 1, binop);
	// inclusive scan does not need the initial value, hence compile error.
	for (unsigned ii = 0; ii < sizedata; ++ii) {
		std::cout << data[ii] << " ";
	}
	std::cout << std::endl;
	return 0;
}
