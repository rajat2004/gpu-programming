#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>

#define N		1024
#define BLOCKSIZE	64

__device__ volatile unsigned k2counter;

__global__ void K2init() {
	k2counter = 0;
}
__global__ void K2() {
	printf("This is before: %d\n", id);

	// global barrier start
	__syncthreads();	// synchronized with all the threads in this block.

	if (threadIdx.x == 0) {	// representative
		atomicInc((unsigned *)&k2counter, gridDim.x + 1);
		while (k2counter != gridDim.x)
			;
	}
	__syncthreads();
	// global barrier end

	printf("This is after the global barrier: %d\n", id);
}
int main() {
	K2init<<<1, 1>>>();
	K2<<<N / BLOCKSIZE, BLOCKSIZE>>>();
	hipDeviceSynchronize();

	return 0;
}
