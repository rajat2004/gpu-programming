#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void K(int *a, int N) {
	#pragma unroll 2
	for (unsigned ii = 0; ii < N; ++ii) {
		a[ii] = ii + 1;
	}
}
int main() {
	int *a, N = 32;
	hipMalloc(&a, N * sizeof(int));

	K<<<1, N>>>(a, N);
	hipDeviceSynchronize();

	return 0;
}
