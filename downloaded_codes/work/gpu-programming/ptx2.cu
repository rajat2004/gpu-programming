#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void K() {
	printf("in K %d\n", blockDim.x);
}
int main() {
	K<<<1, 1>>>();
	hipDeviceSynchronize();

	return 0;
}
