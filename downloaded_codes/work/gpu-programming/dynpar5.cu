#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void Child(int parent) {
	printf("\tparent %d, child %d\n", parent, threadIdx.x + blockIdx.x * blockDim.x);
}
__global__ void Parent() {
	unsigned id = threadIdx.x + blockIdx.x * blockDim.x;
	printf("parent %d\n", id);
	hipStream_t ss;
	hipStreamCreateWithFlags(&ss, hipStreamNonBlocking);
	Child<<<2, 2, 0, ss>>>(id);
	hipDeviceSynchronize();
}
int main() {
	Parent<<<3, 4>>>();
	hipDeviceSynchronize();

	return 0;
}
