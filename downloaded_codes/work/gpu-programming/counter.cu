#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void alloutputs(int *counter) {
    int oldc = atomicAdd(counter, 1);
    if (*counter == 34) printf("%d\n", oldc);
}
int main() {
     int *counter, hcounter = 0;
    hipMalloc(&counter, sizeof(int));
    hipMemcpy(counter, &hcounter, sizeof(int), hipMemcpyHostToDevice);
    alloutputs<<<1, 34>>>(counter);
    hipDeviceSynchronize();
    return 0;
}
