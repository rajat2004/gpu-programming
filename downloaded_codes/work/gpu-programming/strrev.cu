#include <stdio.h>
#include <hip/hip_runtime.h>
#include <string.h>

__global__ void mystrrev(char *str, unsigned lenstr) {
	//if (threadIdx.x < lenstr / 2) {
		char c = str[threadIdx.x];
		str[threadIdx.x] = str[lenstr - threadIdx.x - 1];
		str[lenstr - threadIdx.x - 1] = c;
		printf("%d %c\n", threadIdx.x, c);
	//}
}

int main() {
	char hoststr[100] = "Hello World!";
	char *str;

	unsigned len = strlen(hoststr);
	hipMalloc(&str, sizeof(char) * (len + 1));
	hipMemcpy(str, hoststr, sizeof(char) * (len + 1), hipMemcpyHostToDevice);
	puts(hoststr);
	mystrrev<<<1, len>>>(str, len);
	hipMemcpy(hoststr, str, sizeof(char) * (len + 1), hipMemcpyDeviceToHost);
	puts(hoststr);
	return 0;
}
