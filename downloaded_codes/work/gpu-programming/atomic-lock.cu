#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ int lockvar;
__global__ void k1() {
	while (atomicCAS(&lockvar, 0, 1))
		;
	printf("Block %d, Thread %d is executing critical section.\n", blockIdx.x, threadIdx.x);
	lockvar = 0;
}
int main() {
	hipMemset(&lockvar, 0, sizeof(int));	// lock initialization.
	k1<<<64, 1>>>();
	//k1<<<2, 32>>>();	// This doesn't work.
	hipDeviceSynchronize();

	return 0;
}
