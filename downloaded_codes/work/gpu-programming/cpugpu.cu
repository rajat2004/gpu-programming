#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printk(int *counter) {
	++*counter;
	printf("\t%d\n", *counter);
}
int main() {
	int hcounter = 0, *counter;

	hipMalloc(&counter, sizeof(int));
	do {
		printf("%d\n", hcounter);
		hipMemcpy(counter, &hcounter, sizeof(int), hipMemcpyHostToDevice);
		printk <<<1, 1>>>(counter);
		hipMemcpy(&hcounter, counter, sizeof(int), hipMemcpyDeviceToHost);
	} while (++hcounter < 10);
	return 0;
}
