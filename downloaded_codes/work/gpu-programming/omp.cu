#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime.h>

__global__ void K() {
	printf("in K: %d\n", threadIdx.x);
}
// Compiler as: nvcc -Xcompiler -fopenmp -lgomp omp.cu
int main() {
	omp_set_num_threads(4);
	#pragma omp parallel
	{
		K<<<1, 1>>>();
		hipDeviceSynchronize();
	}

	return 0;
}
