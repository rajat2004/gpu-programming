#include<stdio.h>
#include<hip/hip_runtime.h>

#define N		1024
#define BLOCKSIZE	64

__device__ unsigned binary[N];

__global__ void K() {
	unsigned id = blockDim.x * blockIdx.x + threadIdx.x;
	binary[id] = id;
	__syncthreads();
	if (binary[N-1 - id] != N-1 - id)
		printf("Error: There is no global barrier.\n");
}

int main() {
	K<<<N / BLOCKSIZE, BLOCKSIZE>>>();
	hipDeviceSynchronize();

	return 0;
}
