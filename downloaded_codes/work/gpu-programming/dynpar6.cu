#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

__device__ int value = 5;
__global__ void child() {
	printf("in child %d\n", value);
}
__device__ void devfun() {
	value = value + 2;
	child<<<1, 2>>>();
}
__global__ void parent() {
	devfun();
	value = 4;
	hipDeviceSynchronize();
}
int main() {
	parent<<<1, 2>>>();
	hipDeviceSynchronize();
}



