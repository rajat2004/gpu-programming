#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void printk(int *counter) {
	do {
		while (*counter % 2)
			;
		++*counter;
		//__threadfence_system();
		printf("\t%d\n", *counter);
	} while (*counter < 10);
}
int main() {
	int *counter;

	hipHostAlloc(&counter, sizeof(int), 0);
	//cudaHostAlloc(&counter, sizeof(int), cudaHostAllocMapped);
	printk <<<1, 1>>>(counter);

	do {
		printf("%d\n", *counter);
		//fflush(stdout);
		while (*counter % 2 == 0)
			;
		++*counter;
		//__threadfence_system();
	} while (*counter < 10);

	hipHostFree(counter);
	return 0;
}
