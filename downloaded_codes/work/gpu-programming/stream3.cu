#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void K1() {
	unsigned num = 0;
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	for (unsigned ii = 0; ii < id; ++ii)
		num += ii;
	printf("K1: %d\n", threadIdx.x);
}
__global__ void K2() {
	unsigned num = 0;
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	for (unsigned ii = 0; ii < id; ++ii)
		num += ii;
	__syncthreads();
	printf("K2: %d\n", threadIdx.x);
}
__global__ void K3() {
	printf("\tK3\n");
}
int main() {
	int *ptr;

	hipStream_t s1, s2, s3;
	hipStreamCreate(&s1);
	hipStreamCreate(&s2);
	hipStreamCreate(&s3);

	K1<<<32, 32, 0, s1>>>();
	hipHostAlloc(&ptr, sizeof(int), 0);
	K2<<<1, 1024, 0, s2>>>();
	K3<<<1, 32, 0, s3>>>();
	hipDeviceSynchronize();

	return 0;
}
