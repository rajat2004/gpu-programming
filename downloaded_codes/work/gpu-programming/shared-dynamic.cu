#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dynshared(int sz) {
	extern __shared__ int s[];
	if (threadIdx.x < sz) s[threadIdx.x] = threadIdx.x;
	__syncthreads();
	if (threadIdx.x < sz && threadIdx.x % 2) printf("%d\n", s[threadIdx.x]);
}
int main() {
	int sz;
	scanf("%d", &sz);
	dynshared<<<1, 32, sz * sizeof(int)>>>(sz);
	hipDeviceSynchronize();

	return 0;
}

