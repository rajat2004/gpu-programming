#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dynshared(int sz) {
	extern __shared__ int s[];
	if (threadIdx.x < sz) s[threadIdx.x] = threadIdx.x;
	__syncthreads();
	if (threadIdx.x < sz && threadIdx.x % 2) printf("%d\n", s[threadIdx.x]);
}
int main() {
	int sz;
	scanf("%d", &sz);
	dynshared<<<1, 32, sz * sizeof(int)>>>(sz);
	hipDeviceSynchronize();
    
    // cudaError_t err = cudaGetLastError();
    // printf("error=%d, %s, %s\n", err, cudaGetErrorName(err), cudaGetErrorString(err));

	return 0;
}

