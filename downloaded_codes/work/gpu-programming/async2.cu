#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void K1(int *dst, int nelem) {
	printf("\t%d\n", dst[nelem - 1]);
}
int main() {
	int nbytes = (1 << 30);
	int nelem = nbytes / sizeof(int);
	//int *src = (int *)malloc(nbytes);
	int *src; hipHostAlloc(&src, nbytes, 0);
	src[nelem - 1] = 523;
	int *dst;
	hipMalloc(&dst, nbytes);
for (unsigned ii = 0; ii < 100; ++ii) {
	printf("iteration1 %d\n", ii);
	hipMemcpyAsync(dst, src, nbytes, hipMemcpyHostToDevice);
	printf("iteration2 %d\n", ii);
	K1<<<1, 1>>>(dst, nelem);
	printf("iteration3 %d\n", ii);
	hipDeviceSynchronize();
	hipMemcpy(dst, dst+5, 1, hipMemcpyDeviceToDevice);
	--src[nelem - 1];
}
	return 0;
}
