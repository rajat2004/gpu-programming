#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 20
struct node {
	struct node *next;
	int data;
};

struct node *createNode(int ii) {
	struct node *nn = (struct node *)malloc(sizeof(struct node));
	nn->data = ii;
	nn->next = NULL;

	return nn;
}
struct node *createList() {
	struct node *head = NULL;

	for (int ii = 20; ii > 0; --ii) {
		struct node *nn = createNode(ii);
		nn->next = head;
		head = nn;
	}
	return head;
}
__device__ __host__ void printList(struct node *head) {
	if (head) {
		printf("%d ", head->data);
		printList(head->next);
	} else {
		printf("\n");
	}
}
__global__ void printListGPU(struct node *head) {
	printList(head);
}
struct node *copyNode(struct node *nn) {
	struct node *nngpu;
	hipMalloc(&nngpu, sizeof(struct node));
	hipMemcpy(nngpu, nn, sizeof(struct node), hipMemcpyHostToDevice);
	return nngpu;
}
struct node *copyList(struct node *head) {
	if (!head) return NULL;

	struct node nn;
	nn.next = copyList(head->next);
	nn.data = head->data;
	return copyNode(&nn);
}
int main() {
	struct node *head = createList();
	struct node *gpuhead = copyList(head);

	printList(head);
	printListGPU<<<1, 1>>>(gpuhead);
	hipDeviceSynchronize();

	return 0;
}
