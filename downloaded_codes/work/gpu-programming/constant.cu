#include <hip/hip_runtime.h>
#include <stdio.h>

__constant__ unsigned meta[1];

__global__ void dkernel(unsigned *data) {
	data[threadIdx.x] = meta[0];
}
__global__ void print(unsigned *data) {
	printf("%d %d\n", threadIdx.x, data[threadIdx.x]);
}
int main() {

	unsigned hmeta = 10;
	hipMemcpyToSymbol(HIP_SYMBOL(meta), &hmeta, sizeof(unsigned));
	unsigned *data;
	hipMalloc(&data, 32 * sizeof(unsigned));
	dkernel<<<1, 32>>>(data);
	hipDeviceSynchronize();
	print<<<1, 32>>>(data);
	hipDeviceSynchronize();
	return 0;
}
