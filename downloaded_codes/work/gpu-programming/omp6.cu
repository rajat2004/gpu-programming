#include <stdio.h>
#include <omp.h>
#include <hip/hip_runtime.h>

#define N 10

__device__ __host__ void fun(int *a, int ii) {
	a[ii] = ii + 1;
}
__global__ void K(int *a) {
	fun(a, threadIdx.x);
}
int main() {
	int *a;
	hipHostAlloc(&a, sizeof(int) * N, 0);
	K<<<1, N/2>>>(a);

	#pragma omp parallel for
	for (int ii = N/2; ii < N; ++ii)
		fun(a, ii);
	hipDeviceSynchronize();

	for (int ii = 0; ii < N; ++ii)
		printf("a[%d] = %d\n", ii, a[ii]);
	return 0;
}
