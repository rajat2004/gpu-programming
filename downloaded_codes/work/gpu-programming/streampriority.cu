#include <stdio.h>
#include <hip/hip_runtime.h>

int main() {
// get the range of stream priorities for this device
int priority_high, priority_low;
hipDeviceGetStreamPriorityRange(&priority_low, &priority_high);
// create streams with highest and lowest available priorities
hipStream_t st_high, st_low;
hipStreamCreateWithPriority(&st_high, hipStreamNonBlocking, priority_high);
hipStreamCreateWithPriority(&st_low, hipStreamNonBlocking, priority_low);
printf("%d -- %d\n", priority_low, priority_high);
}
