#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void hello() {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	//if (id == 2047)
		printf("my id is %d.\n", id);
}
int main() {
	dim3 block(1024, 1, 1);
	hello<<<2, block>>>();
	hipDeviceSynchronize();
	return 0;
}
