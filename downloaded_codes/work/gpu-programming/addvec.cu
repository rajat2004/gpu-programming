#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdlib.h>

#define BLOCKSIZE	1024

__global__ void addvec(int *a, int *b, int *c, int n) {
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < n)
		c[id] = a[id] + b[id];
}
void init(int *a, int *b, int n) {
	srand(time(NULL));
	for (int ii = 0; ii < n; ++ii) {
		a[ii] = rand() % 100;
		b[ii] = rand() % 100;
	}
}
int main() {
	int *a, *b, *c;
	int *da, *db, *dc;

	int n = 10000;
	dim3 block(BLOCKSIZE, 1, 1);
	dim3 grid(ceil((float)n/BLOCKSIZE), 1, 1);
	printf("number of blocks = %d\n", ceil((float)n/BLOCKSIZE));

	a = (int *)malloc(n*sizeof(int));
	b = (int *)malloc(n*sizeof(int));
	c = (int *)malloc(n*sizeof(int));

	hipMalloc(&da, n*sizeof(int));
	hipMalloc(&db, n*sizeof(int));
	hipMalloc(&dc, n*sizeof(int));

	init(a, b, n);

	hipMemcpy(da, a, n*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(db, b, n*sizeof(int), hipMemcpyHostToDevice);

	addvec<<<grid, block>>>(da, db, dc, n);
	hipDeviceSynchronize();

	hipMemcpy(c, dc, n*sizeof(int), hipMemcpyDeviceToHost);

	for (int ii = 0; ii < n; ++ii) {
		printf("%d ", c[ii]);
	}
	printf("\n");
	return 0;
}
