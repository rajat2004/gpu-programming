#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE	1024

__global__ void dkernel() {
	__shared__ unsigned s;

	if (threadIdx.x == 0) s = 0;
	__syncthreads();

	if (threadIdx.x == 1) s += 1;
	__syncthreads();

	if (threadIdx.x == 100) s += 2;
	__syncthreads();

	if (threadIdx.x == 0) printf("s=%d\n", s);
}
int main() {
	int i;
	for (i = 0; i < 10; ++i) {
		dkernel<<<2, BLOCKSIZE>>>();
		hipDeviceSynchronize();
	}
}
