#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void Child(int father) {
	printf("Parent %d -- Child %d\n", father, threadIdx.x);
}
__global__ void Parent() {
	printf("Parent %d\n", threadIdx.x);
	Child<<<1, 5>>>(threadIdx.x);
}
int main() {
	Parent<<<1, 3>>>();
	hipDeviceSynchronize();

	return 0;
}
