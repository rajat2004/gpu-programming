#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>

class dreference {
public:
	dreference(int *memloc) {
		this->memloc = memloc;
	}
	int operator ()() {
		return getval();
	}
	int operator = (int newval) {
		//printf("Writing %d at %p\n", newval, memloc);
		hipMemcpy(memloc, &newval, sizeof(int), hipMemcpyHostToDevice);
		return newval;	// can return self-reference to allow cascaded =.
	}
	int getval() { 
		int val;
		hipMemcpy(&val, memloc, sizeof(int), hipMemcpyDeviceToHost);
		return val; 
	}
private:
	int *memloc;
};

class dvector {
public:
	dvector(unsigned size);
	~dvector();
	dreference operator [](unsigned ii);
	void print();
private:
	int *arr;
	int size;
};

dvector::dvector(unsigned size) {
	hipMalloc(&arr, size * sizeof(int));
	this->size = size;
	//printf("arr points to %p\n", arr);
}
dvector::~dvector() {
	hipFree(arr);
	arr = NULL;
}
dreference dvector::operator [](unsigned ii) {
	return dreference(arr + ii);
}
void dvector::print() {
	int aval;
	for (int ii = 0; ii < size; ++ii) {
		hipMemcpy(&aval, arr + ii, sizeof(int), hipMemcpyDeviceToHost);
		std::cout << aval << ", ";
	}
	std::cout << std::endl;
}
std::ostream & operator <<(std::ostream &os, dreference dd) {
	return os << dd.getval();
}
int main() {
	dvector dv(10);
	dv[0] = 1;
	dv[1] = 2;
	dv[5] = 2;

	std::cout << dv[0] << ", " << dv[1] << std::endl;
	dv.print();

	return 0;
}
