#include <stdio.h>
#include <hip/hip_runtime.h>

#define N	10

__host__ __device__ void fun(int *arr) {
	for (unsigned ii = 0; ii < N; ++ii)
		++arr[ii];
}
__global__ void dfun(int *arr) {
	fun(arr);
}
__global__ void dprint(int *arr, int x);
__host__ __device__ void print(int *arr, int x) {
	for (unsigned ii = 0; ii < N; ++ii)
		printf("%d, ", arr[ii]);

	printf("\n");
	dprint<<<1, 5>>>(arr, ++x);
}
__global__ void dprint(int *arr, int x = 0) {
	if (x < 1) print(arr, x);
}
int main() {
	int arr[N], *darr;

	hipMalloc(&darr, N * sizeof(int));

	for (unsigned ii = 0; ii < N; ++ii)
		arr[ii] = ii;
	hipMemcpy(darr, arr, N * sizeof(int), hipMemcpyHostToDevice);

	fun(arr);
	dfun<<<1, 1>>>(darr);
	hipDeviceSynchronize();

	print(arr, -1);
	dprint<<<1, 1>>>(darr);
	hipDeviceSynchronize();

	return 0;
}
