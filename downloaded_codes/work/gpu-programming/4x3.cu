#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void K() {
	printf("%d\n", threadIdx.x + threadIdx.y);
}
int main() {
	dim3 block(3, 4);
	K<<<1, block>>>();
	hipDeviceSynchronize();

	return 0;
}
