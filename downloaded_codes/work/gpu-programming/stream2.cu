#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 2

__global__ void K(int *out, int *in, int size) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	out[id] = in[id] * in[id];
}
int main() {
	hipStream_t stream[N];
	for (unsigned ii = 0; ii < N; ++ii)
		hipStreamCreate(&stream[ii]);

	int *hptr, *dinptr, *doutptr;
	unsigned nbytesperstream = (1<<10);
	unsigned nbytes = N * nbytesperstream;
	hipHostAlloc(&hptr, nbytes, 0);
	hipMalloc(&dinptr, nbytes);
	hipMalloc(&doutptr, nbytes);

	for (unsigned ii = 0; ii < N; ++ii) {
		hipMemcpyAsync(dinptr + ii * nbytesperstream, hptr + ii * nbytesperstream, nbytesperstream, hipMemcpyHostToDevice, stream[ii]);
		K<<<nbytesperstream / 512, 512, 0, stream[ii]>>>(doutptr + ii * nbytesperstream, dinptr + ii * nbytesperstream, nbytesperstream);
		hipMemcpyAsync(hptr + ii * nbytesperstream, doutptr + ii * nbytesperstream, nbytesperstream, hipMemcpyDeviceToHost, stream[ii]);
	}

	return 0;
}
