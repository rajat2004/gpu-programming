#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "mytime.h"

#define N 4096
// This program works for ODD value of N as well

void fill_values(int *array, int n) {
	time_t t;
	srand((unsigned) time(&t));
	
    for(int j=0; j<n; j++) {
		//array[j]=j;
		if(j % 2 == 0)  
			array[j] = rand() % 200;  
		else{	
			//array[j] = j - j * rand() % (rand() * 200);  
			array[j] = rand() % 100 + rand() % 100;  	
		}
	}
	
}

void printValues(int *a , int n){

	int sum=0;
	for(int i=0;i <n; i++){
		//printf("%4d", a[i]);
		sum += a[i];
	}
	//printf("\n");
	//printf("SUM: %d\n", sum);
}


__global__ void dk(int *a, int n, int iteration){
	unsigned id = threadIdx.x + blockIdx.x * blockDim.x;
	if(id < n){
		unsigned index = id * (1 << (iteration+1));
		//unsigned index = id * (int)pow(2.0, iteration+1);
		unsigned shift_index = (1 << iteration);
		//unsigned shift_index = (int)pow(2.0, iteration);
		//if( n % 2 == 1 && id == n-1 ){
		//	a[index] = a[index] + a[ index + shift_index ] +a[index + 2 * shift_index ];
			//printf("a[%d] = a[%d] + a[%d] + a[%d] = %d \n",index,index, index + shift_index ,index + 2 * shift_index , a[index]);
		//}
		//else{
			a[index] = a[index] + a[ index + shift_index];
			//printf("a[%d] = a[%d] + a[%d] = %d\n",index, index, index + shift_index , a[index]);
		//}
	}
	//__syncthreads();

}
int main(int argc, char** argv){
	double start, end;
	
	unsigned bytes = sizeof(int) * N;
	//~ unsigned sumbytes = sizeof(int) ;
	
	int *a 	= (int *) malloc (bytes);
	//~ int *sum= (int *) malloc (sumbytes);
	fill_values(a,N); // fills random values
	
	//CPUTimer cputimer;
    //cputimer.Start();
	start = rtclock();
  
	printValues(a,N); // prints and finds cpu sum as well.
	
	//cputimer.Stop();
	end = rtclock();
	printtime("Sequential time: ", start, end);
	//printf("The sequential code ran in %f ms\n", cputimer.Elapsed()*1000);
	
	int  *da;
	//~ int *dsum; // removing it as a[0] stores the final result
	hipMalloc(&da, bytes);
	//~ hipMalloc(&dsum, sumbytes);
	
	hipMemset(da, 0,bytes);
	//~ hipMemset(dsum, 0,sumbytes);
	
	hipMemcpy(da,a,bytes, hipMemcpyHostToDevice);
	
	unsigned numThreads = 1024;
	//GPUTimer gputimer;
    //gputimer.Start();
  	start = rtclock();
	for(int i = N/2, j=0; i > 0; j++,i=i/2)	 {
		dk<<< (ceil((float)i/numThreads)) , numThreads>>>(da, i, j);
		//hipDeviceSynchronize();
	}
		
	//dk<<< 1, i>>>(da, i, j);
	//~ dk<<< 1, N/2>>>(da, N, dsum, 0);
	//~ dk<<< 1, N/4>>>(da, N, dsum, 1);
	//~ dk<<< 1, N/8>>>(da, N, dsum, 2);
	//~ printValues(a,N);
	
	//gputimer.Stop();
	end = rtclock();
	//printf("The Parallel code ran in %f ms\n", gputimer.Elapsed()*1000);
	printtime("Parallel time: ", start, end);
	
	
	//~ hipMemcpy(sum,dsum,sumbytes, hipMemcpyDeviceToHost);
	hipMemcpy(a,da, bytes, hipMemcpyDeviceToHost);
	printf("Gpu sum %d\n", a[0]);

	hipFree(da);
	//~ hipFree(dsum);
	return 0;
}
