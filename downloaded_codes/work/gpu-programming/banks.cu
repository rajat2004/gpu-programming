#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include "mytime.h"


__global__ void bankcheck() {
	__shared__ unsigned s[1024];
	s[1 * threadIdx.x] = threadIdx.x;
}
__global__ void bankcheck2() {
	__shared__ unsigned s[1024];
	s[32 * threadIdx.x] = threadIdx.x;
}
int main() {
	int ii;
	double start, end;

	bankcheck<<<1, 32>>>();	// dummy for warmup.
	hipDeviceSynchronize();

	start = rtclock();
	for (ii = 0; ii < 1000; ++ii) {
		bankcheck<<<1, 32>>>();
		hipDeviceSynchronize();
	}
	end = rtclock();
	printtime("bank consecutive: ", start, end);

	start = rtclock();
	for (ii = 0; ii < 1000; ++ii) {
		bankcheck2<<<1, 32>>>();
		hipDeviceSynchronize();
	}
	end = rtclock();
	printtime("bank strided: ", start, end);
	return 0;
}
