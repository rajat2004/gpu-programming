#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#define N 10
__global__ void f() {
	printf("%d\n", threadIdx.x);
}
int main() {
	f<<<1, N>>>();
	hipDeviceSynchronize();
	return 0;
}
