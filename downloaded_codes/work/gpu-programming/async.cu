#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void K1() {
	unsigned sum = 0;
	if (blockIdx.x == 0 && threadIdx.x == 0)
		printf("K1 before\n");
	for (unsigned ii = 0; ii < 1000; ++ii) {
		sum += ii;
	}
	if (blockIdx.x == 0 && threadIdx.x == 0)
		printf("K1 after\n");
}
__global__ void K2() {
	printf("in K2\n");
}
int main() {
	printf("on CPU\n");
	K1<<<10, 32, 0, 0>>>();
	K2<<<1, 1>>>();
	hipDeviceSynchronize();
	printf("on CPU\n");
	return 0;
}
