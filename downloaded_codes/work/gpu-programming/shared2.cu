#include <stdio.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE	1024

__global__ void dkernel() {
	__shared__ unsigned data[12*1024];
	data[threadIdx.x] = threadIdx.x;
}
int main() {
	hipFuncSetCacheConfig(reinterpret_cast<const void*>(dkernel), hipFuncCachePreferL1);
	//cudaFuncSetCacheConfig(dkernel, cudaFuncCachePreferShared);
	dkernel<<<1, BLOCKSIZE>>>();
	hipDeviceSynchronize();
}
