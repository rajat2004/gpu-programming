#include <stdio.h>
#include <hip/hip_runtime.h>
__host__ __device__ void dhfun() {
	printf("I can run on both CPU and GPU.\n");
}
__device__ unsigned dfun(unsigned *vector, unsigned vectorsize, unsigned id) {
	if (id == 0) dhfun();
	if (id < vectorsize) {
		vector[id] = id;
		return 1;
	} else {
		return 0;
	}
}
__global__ void dkernel(unsigned *vector, unsigned vectorsize) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	dfun(vector, vectorsize, id);
}
__host__ void hostfun() {
	printf("I am simply like another function running on CPU. Calling dhfun\n");
	dhfun();
}
#define BLOCKSIZE	1024
int main(int nn, char *str[]) {
	unsigned N = atoi(str[1]);
	unsigned *vector, *hvector;
	hipMalloc(&vector, N * sizeof(unsigned));
	hvector = (unsigned *)malloc(N * sizeof(unsigned));

	unsigned nblocks = ceil((float)N / BLOCKSIZE);
	printf("nblocks = %d\n", nblocks);

    	dkernel<<<nblocks, BLOCKSIZE>>>(vector, N);
	hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);
	for (unsigned ii = 0; ii < N; ++ii) {
		printf("%4d ", hvector[ii]);
	}
	printf("\n");
	hostfun();
	dhfun();
    	return 0;
}
