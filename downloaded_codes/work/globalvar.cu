#include <stdio.h>
#include <hip/hip_runtime.h>
unsigned int N = 32;
// #define N 32

__global__ void dkernel(int m) {
	unsigned id = threadIdx.x;
    // unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
    printf("id = %d, N = %d, m = %d.\n", id, N, m);
}
int main() {
	unsigned id = 1;
	dkernel<<<N, id>>>(N);
	hipDeviceSynchronize();
    return 0;
}
