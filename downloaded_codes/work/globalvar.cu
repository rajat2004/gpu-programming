#include <stdio.h>
#include <hip/hip_runtime.h>
unsigned int N = 32;

__global__ void dkernel(int m) {
	unsigned id = threadIdx.x;
    printf("id = %d, N = %d, m = %d.\n", N, m);
}
int main() {
	unsigned id = 1;
	dkernel<<<N, id>>>(N);
	hipDeviceSynchronize();
    return 0;
}
