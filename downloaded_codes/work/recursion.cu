#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
__device__ unsigned dfun(unsigned id) {
	printf("%d\n", id);
	if (id > 10 && id < 15) return dfun(id+1);
	else return 0;
}
__global__ void dkernel(unsigned n) {
	dfun(n);
	
}

#define BLOCKSIZE	256
int main(int nn, char *str[]) {
	unsigned N = atoi(str[1]);
    	dkernel<<<1, BLOCKSIZE>>>(N);
	hipDeviceSynchronize();
    	return 0;
}
