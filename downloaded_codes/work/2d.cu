#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(unsigned *matrix) {
	unsigned id = threadIdx.x * blockDim.y + threadIdx.y;
	matrix[id] = id;
}
#define N	5
#define M	6
int main() {
	dim3 block(N, M, 1);
	unsigned *matrix, *hmatrix;
	hipMalloc(&matrix, N * M * sizeof(unsigned));
	hmatrix = (unsigned *)malloc(N * M * sizeof(unsigned));
    	dkernel<<<1, block>>>(matrix);
	hipMemcpy(hmatrix, matrix, N * M * sizeof(unsigned), hipMemcpyDeviceToHost);
	for (unsigned ii = 0; ii < N; ++ii) {
		for (unsigned jj = 0; jj < M; ++jj) {
			printf("%2d ", hmatrix[ii * M + jj]);
		}
		printf("\n");
	}
    return 0;
}
