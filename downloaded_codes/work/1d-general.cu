#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(unsigned *vector, unsigned vectorsize) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id < vectorsize) vector[id] = id;
}
#define BLOCKSIZE	1024
int main(int nn, char *str[]) {
	unsigned N = atoi(str[1]);
	unsigned *vector, *hvector;
	hipMalloc(&vector, N * sizeof(unsigned));
	hvector = (unsigned *)malloc(N * sizeof(unsigned));

	unsigned nblocks = ceil((float)N / BLOCKSIZE);
	printf("nblocks = %d\n", nblocks);

    	dkernel<<<nblocks, BLOCKSIZE>>>(vector, N);
	hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);
	for (unsigned ii = 0; ii < N; ++ii) {
		printf("%4d ", hvector[ii]);
		if (ii % 1000 == 0) printf("\n");
	}
    	return 0;
}
