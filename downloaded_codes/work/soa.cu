#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>
#define N 1024
struct nodeAOS {
	int a;
	double b;
	char c;
} *allnodesAOS;
struct nodeSOA {
	int *a;
	double *b;
	char *c;
} allnodesSOA;
__global__ void dkernelaos(struct nodeAOS *allnodesAOS) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	allnodesAOS[id].a = id;
	allnodesAOS[id].b = 0.0;
	allnodesAOS[id].c = 'c';
}
__global__ void dkernelsoa(int *a, double *b, char *c) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	a[id] = id;
	b[id] = 0.0;
	c[id] = 'd';
}

double rtclock() {
  struct timezone Tzp;
  struct timeval Tp;
  int stat;
  stat = gettimeofday(&Tp, &Tzp);
  if (stat != 0) printf("Error return from gettimeofday: %d", stat);
  return(Tp.tv_sec + Tp.tv_usec * 1.0e-6);
}
void printtime(const char *str, double starttime, double endtime) {
	printf("%s%3f seconds\n", str, endtime - starttime);
}

#define BLOCKSIZE	1024
int main(int nn, char *str[]) {
	hipMalloc(&allnodesAOS, N * sizeof(struct nodeAOS));
	hipMalloc(&allnodesSOA.a, N * sizeof(int));
	hipMalloc(&allnodesSOA.b, N * sizeof(double));
	hipMalloc(&allnodesSOA.c, N * sizeof(char));

	unsigned nblocks = ceil((float)N / BLOCKSIZE);

	double starttime = rtclock();
    	dkernelaos<<<nblocks, BLOCKSIZE>>>(allnodesAOS);
	hipDeviceSynchronize();
	double endtime = rtclock();
	printtime("AoS time: ", starttime, endtime);

	starttime = rtclock();
    	dkernelsoa<<<nblocks, BLOCKSIZE>>>(allnodesSOA.a, allnodesSOA.b, allnodesSOA.c);
	hipDeviceSynchronize();
	endtime = rtclock();
	printtime("SoA time: ", starttime, endtime);

    	return 0;
}
