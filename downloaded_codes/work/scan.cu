#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>

#define N	8
__global__ void exscan() {
	__shared__ unsigned a[N]; //= {4, 3, 9, 3, 5, 7, 3, 2};
	if (threadIdx.x == 0) {
		a[0] = 4; a[1] = 3; a[2] = 9; a[3] = 3;
		a[4] = 5; a[5] = 7; a[6] = 3; a[7] = 2;
	}
	__syncthreads();
	unsigned n = sizeof(a) / sizeof (*a);

	__syncthreads();
	for (int off = 0; off < n; off *= 2) {
		if (threadIdx.x > off) {
			a[threadIdx.x] += a[threadIdx.x - off];
		}
		__syncthreads();
	}
	__syncthreads();
	if (threadIdx.x == 0) {
		for (unsigned ii = 0; ii < n; ++ii)
			printf("%d ", a[ii]);
		printf("\n");
	}
}
int main() {
	//hipSetDevice(5);
	exscan<<<1, 32>>>();
	hipDeviceSynchronize();
	return 0;
}

