#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(unsigned *vector, unsigned vectorsize) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	vector[id] = id;
	__syncthreads();

	if (id < vectorsize - 1 && vector[id + 1] != id + 1) printf("syncthreads does not work.\n");
}
#define BLOCKSIZE	1000
#define N		BLOCKSIZE
int main(int nn, char *str[]) {
	unsigned *vector, *hvector;
	hipMalloc(&vector, N * sizeof(unsigned));
	hvector = (unsigned *)malloc(N * sizeof(unsigned));

    	dkernel<<<100, BLOCKSIZE>>>(vector, N);
	hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);
	for (unsigned ii = 0; ii < N; ++ii) {
		printf("%4d ", hvector[ii]);
	}
	printf("\n");
    	return 0;
}
