#include <stdio.h>
#include <hip/hip_runtime.h>
__global__ void dkernel(unsigned *vector, unsigned vectorsize) {
	unsigned id = blockIdx.x * blockDim.x + threadIdx.x;
	if (id % 2) vector[id] = id;
	else vector[id] = vectorsize * vectorsize;
}
#define BLOCKSIZE	10
#define N		BLOCKSIZE
int main(int nn, char *str[]) {
	unsigned *vector, *hvector;
	hipMalloc(&vector, N * sizeof(unsigned));
	hvector = (unsigned *)malloc(N * sizeof(unsigned));

	unsigned nblocks = ceil((float)N / BLOCKSIZE);

    	dkernel<<<nblocks, BLOCKSIZE>>>(vector, N);
	hipMemcpy(hvector, vector, N * sizeof(unsigned), hipMemcpyDeviceToHost);
	for (unsigned ii = 0; ii < N; ++ii) {
		printf("%4d ", hvector[ii]);
	}
	printf("\n");
    	return 0;
}
