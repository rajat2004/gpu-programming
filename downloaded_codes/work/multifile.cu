#include "hip/hip_runtime.h"
/* compile as:
	nvcc -c multifile.cu
	g++ multifile-cfunction.c multifile.o -L/usr/local/cuda/lib64/ -lcuda -lcudart

  compiling as below results in linking error not finding cfunction.
	nvcc multifile.cu multifile-cfunction.c
*/
#include <stdio.h>
#include <hip/hip_runtime.h>

__global__ void dkernel(unsigned n) {
	printf("in dkernel %d\n", blockIdx.x * blockDim.x + threadIdx.x);
}
void cfunction();

#define BLOCKSIZE	32
int main() {
	unsigned N = BLOCKSIZE;
    	dkernel<<<1, BLOCKSIZE>>>(N);
	hipDeviceSynchronize();
	cfunction();
    	return 0;
}
