#include<stdio.h>
#include<hip/hip_runtime.h>

void print_arr(int* arr, int size) {
    for(int i=0; i<size; i++)
        print("%d ", arr[i]);
    print("\n");
}


int main() {
    int *vec1, *vec2, *dvec1, *dvec2;
    int N;
    scanf("%d", &N);

    // Two vectors of same size
    hipMalloc(&dvec1, N*sizeof(int));
    hipMalloc(&dvec2, N*sizeof(int));

    vec1 = (int*)malloc(N*sizeof(int));
    vec2 = (int*)malloc(N*sizeof(int));
    return 0;
}
