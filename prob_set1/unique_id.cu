#include<stdio.h>
#include<hip/hip_runtime.h>

__global__ void find_unique_id(int* arr) {
    int bid = (blockIdx.z * gridDim.y * gridDim.x) + (blockIdx.y * gridDim.x) + blockIdx.x;
    int tid = (bid * blockDim.x * blockDim.y * blockDim.z) + (threadIdx.z * blockDim.y * blockDim.x) + (threadIdx.y * blockDim.x) + threadIdx.x;

    arr[tid] = tid;
}

int main() {
    dim3 grid(1,2,3);
    dim3 block(4,5,6);
    int threads = 1*2*3*4*5*6;

    int *arr, *darr;
    arr = (int*)malloc(threads*sizeof(int));
    hipMalloc(&darr, threads*sizeof(int));

    find_unique_id<<<grid, block>>>(darr);
    hipDeviceSynchronize();
    hipMemcpy(arr, darr, threads*sizeof(int), hipMemcpyDeviceToHost);

    for(int i=0; i<threads; i++) 
        printf("%d\n", arr[i]);
    printf("\n");
    return 0;
}
