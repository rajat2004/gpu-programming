#include "hip/hip_runtime.h"
#include<stdio.h>
#include<hip/hip_runtime.h>
#include<sys/time.h>
#include<stdlib.h>

void print_arr(int* arr, int size) {
    for(int i=0; i<size; i++)
        printf("%d ", arr[i]);
    printf("\n");
}

void init(int *arr, int n) {
    srand(time(NULL));
    for(int i=0; i<n; i++)
        arr[i]=rand()%10;
}

__global__ void add_vec(int *a, int *b, int *c, int n1) {
    int tid = blockIdx.x*blockDim.x + threadIdx.x;
    if(tid<n1)
        c[tid] = a[tid] + b[tid];
}

__global__ void kernel() {
    printf("%d\n", threadIdx.x);
}

int main() {
    int *vec1, *vec2, *res, *dvec1, *dvec2, *dvec3;
    int N;
    // scanf("%d", &N);

    // Vectors of same size
    // hipMalloc(&dvec1, N*sizeof(int));
    // hipMalloc(&dvec2, N*sizeof(int));
    // hipMalloc(&dvec3, N*sizeof(int));

    // vec1 = (int*)malloc(N*sizeof(int));
    // vec2 = (int*)malloc(N*sizeof(int));
    // res = (int*)malloc(N*sizeof(int));

    kernel<<<1, 1025>>>();
    hipDeviceSynchronize();

    // check for error
    hipError_t error = hipGetLastError();
    if(error != hipSuccess)
    {
      // print the CUDA error message and exit
      printf("CUDA error: %s\n", hipGetErrorString(error));
      exit(-1);
    }
    return 0;
}
