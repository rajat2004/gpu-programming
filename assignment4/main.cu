#include <stdio.h>
#include <hip/hip_runtime.h>
#include "kernel.h"

int main(int argc, char* argv[]) {
  int N, M;
  FILE* fp = fopen(argv[1], "r");
  fscanf(fp, "%d %d", &N, &M);

  int *arrival_times, *burst_times;
  int **cores_schedules, *cs_lengths;
  int turnaround_time = 0;
  int i;
  
  arrival_times = (int*) malloc(N * sizeof(int));
  burst_times = (int*) malloc(N * sizeof(int));
  cores_schedules = (int**) malloc(M * sizeof(int*));
  cs_lengths = (int*) malloc(M * sizeof(int));

  for(i = 0; i < N; i++) {
	fscanf(fp, "%d %d", &arrival_times[i], &burst_times[i]);
  }

  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);
  float milliseconds = 0;
  hipEventRecord(start,0);

  turnaround_time = schedule(N, M, arrival_times, burst_times, cores_schedules, cs_lengths);

  hipDeviceSynchronize();

  hipEventRecord(stop,0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&milliseconds, start, stop);
  printf("%f\n",milliseconds);

  printf("%d\n", turnaround_time);
  for(int i = 0; i < M; i++){
	for(int j = 0; j < cs_lengths[i]; j++){
	  printf("%d ", cores_schedules[i][j]);
	}
	printf("\n");
  }
}
